#include "hip/hip_runtime.h"
#define TARGET __device__ __host__
#include "Solve_Polynomial.inc"

__device__
void GetInfo_Torus(const Task& task, const sp& K, const sp& L, Info4cuda& info)
{
	int Solve_Polynomial(int d, double *k, double min, double max, double *r);

	double m_R = task.data.torus.R;
	double m_r = task.data.torus.r;

	double	R2 = m_R * m_R;
	double	a = K * K;
	double	b = 2 * (K * L);
	double	c = L * L + R2 - m_r * m_r;

	double	k[20], r[50];
	int		n;

	k[4] = a * a;
	k[3] = 2 * a * b;
	k[2] = 2 * a * c + b * b - 4 * R2 * (K.x * K.x + K.y * K.y);
	k[1] = 2 * b * c - 8 * R2 * (K.x * L.x + K.y * L.y);
	k[0] = c * c - 4 * R2 * (L.x * L.x + L.y * L.y);

	n = Solve_Polynomial(4, k, 0.0, 2000, r);

	if (n == 0 || fabs(r[0]) < 1.0) {
		info.valid = false;
		return;
	}

	double	th;

	double d = m_R - sqrt(L.x * L.x + L.y * L.y);
	d *= d;

	info.isEnter = (m_r * m_r <= d + L.z * L.z);
	
	sp p = K * r[0] + L;
	info.Cross_x = p.x;
	info.Cross_y = p.y;
	info.Cross_z = p.z;
	
	if (p.x == 0.0) {
		info.Vertical_x = 0;
		info.Vertical_y = p.y - ((p.y > 0) ? 1 : -1) * m_R;
		info.Vertical_z = p.z;
	} else {
		th = atan2(p.y , p.x);
		info.Vertical_x = p.x - m_R * cos(th);
		info.Vertical_y = p.y - m_R * sin(th);
		info.Vertical_z = p.z;
	}

	info.Distance = r[0] * sqrt(K * K);

	double x,y,z, phy;

	x = info.Vertical_x;
	y = info.Vertical_y;
	z = info.Vertical_z;

	th = acos(y) / (2 * M_PI); if (x < 0) th = 1 - th;
	phy = acos(z / sqrt(x * x + z * z)) / (2 * M_PI);
	if (x < 0)
		phy = 1 - phy;

	//info.Material = GetPixel(phy, th).getMaterial();
	info.Material = sp(256 * task.nodeInfo.m_Material.Diffuse.r,
					   256 * task.nodeInfo.m_Material.Diffuse.g,
					   256 * task.nodeInfo.m_Material.Diffuse.b).getMaterial();
	info.nodeInfo = task.nodeInfo;

	info.valid = true;
}

