#include "hip/hip_runtime.h"
#pragma once

#include "sp.cuh"
#include "Matrix.cuh"

Sp::Sp( const Matrix& m )
{
	x = m.get_data(1,1);
	y = m.get_data(2,1);
	z = m.get_data(3,1);
}

bool Sp::operator==( const Sp& a ) const
{
	return x == a.x && y == a.y && z == a.z;
}

bool Sp::operator!=( const Sp& a ) const
{
	return x != a.x || y != a.y || z != a.z;
}

Sp&	Sp::operator=( const Sp& a )
{
	x = a.x;
	y = a.y;
	z = a.z;
	return *this;
}

double	Sp::operator*( const Sp& a ) const
{
	return x*a.x+y*a.y+z*a.z;
}

Sp Sp::operator+( const Sp& a) const
{
	return Sp( x+a.x, y+a.y, z+a.z );
}

Sp Sp::operator-( const Sp& a) const
{
	return Sp( x-a.x, y-a.y, z-a.z );
}

Sp Sp::operator-( void ) const
{
	return Sp( -x, -y, -z );
}

Sp Sp::operator*( double k ) const
{
	return Sp( x*k, y*k, z*k );
}

Sp Sp::operator/( double a ) const
{
	return Sp( x/a, y/a, z/a );
}
/*
void Sp::print()
{
	printf( "(%f,%f,%f)\n", x, y, z );
}
*/
Sp	operator*( double k, const Sp& a)
{
	return a*k;
}

Sp	Sp::e() const
{
	double	r = sqrt( x*x+y*y+z*z );

	return Sp( x/r, y/r, z/r );
}

/*
POINT sp::getPOINT(const CSize& size) const
{
	POINT ans;
	if (z > -20) {
		ans.x = (long)(((x / (PERSPECTIVE_RATIO * (z + 20))) + 10) * size.cx / 20);
		ans.y = (long)(((y / (PERSPECTIVE_RATIO * (z + 20))) + 10) * size.cx / 20);
	} else {
		ans.x = (long)(((x / 1E-10) + 10) * size.cx / 20);
		ans.y = (long)(((y / 1E-10) + 10) * size.cx / 20);
	}
	return ans;
}
*/

D3DMATERIAL9 Sp::getMaterial() const
{
	D3DMATERIAL9	mtrl;
	ZeroMemory(&mtrl, sizeof(D3DMATERIAL9));
	mtrl.Diffuse.r = mtrl.Ambient.r = (float)(x / 256);
	mtrl.Diffuse.g = mtrl.Ambient.g = (float)(y / 256);
	mtrl.Diffuse.b = mtrl.Ambient.b = (float)(z / 256);
	mtrl.Diffuse.a = mtrl.Ambient.a = 1.0f;
	return mtrl;
}

Sp::Sp(const D3DMATERIAL9& mtrl)
{
	x = 256 * mtrl.Diffuse.r;
	y = 256 * mtrl.Diffuse.g;
	z = 256 * mtrl.Diffuse.b;
}
