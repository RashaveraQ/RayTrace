#include "hip/hip_runtime.h"
#pragma once

#include <math.h>
#include "sp.cuh"
#include "matrix.cuh"

__device__
sp::sp(const matrix& m)
{
	x = m.get_data(1,1);
	y = m.get_data(2,1);
	z = m.get_data(3,1);
}

__device__
bool sp::operator==( const sp& a ) const
{
	return x == a.x && y == a.y && z == a.z;
}

__device__ 
bool sp::operator!=(const sp& a) const
{
	return x != a.x || y != a.y || z != a.z;
}

__device__ 
sp&	sp::operator=(const sp& a)
{
	x = a.x;
	y = a.y;
	z = a.z;
	return *this;
}

__device__ 
float	sp::operator*(const sp& a) const
{
	return x*a.x+y*a.y+z*a.z;
}

__device__ 
sp sp::operator+(const sp& a) const
{
	return sp( x+a.x, y+a.y, z+a.z );
}

__device__
sp sp::operator-(const sp& a) const
{
	return sp( x-a.x, y-a.y, z-a.z );
}

__device__
sp sp::operator-( void ) const
{
	return sp( -x, -y, -z );
}

__device__
sp sp::operator*(float k) const
{
	return sp( x*k, y*k, z*k );
}

__device__
sp sp::operator/(float a) const
{
	return sp( x/a, y/a, z/a );
}
/*
void sp::print()
{
	printf( "(%f,%f,%f)\n", x, y, z );
}
*/

__device__ 
sp	operator*(float k, const sp& a)
{
	return a*k;
}

__device__
sp	sp::e() const
{
	float	r = sqrt(x*x + y*y + z*z);

	return sp( x/r, y/r, z/r );
}

/*
POINT sp::getPOINT(const CSize& size) const
{
	POINT ans;
	if (z > -20) {
		ans.x = (long)(((x / (PERSPECTIVE_RATIO * (z + 20))) + 10) * size.cx / 20);
		ans.y = (long)(((y / (PERSPECTIVE_RATIO * (z + 20))) + 10) * size.cx / 20);
	} else {
		ans.x = (long)(((x / 1E-10) + 10) * size.cx / 20);
		ans.y = (long)(((y / 1E-10) + 10) * size.cx / 20);
	}
	return ans;
}
*/

__device__
D3DMATERIAL9 sp::getMaterial() const
{
	D3DMATERIAL9	mtrl;
	ZeroMemory(&mtrl, sizeof(D3DMATERIAL9));
	mtrl.Diffuse.r = mtrl.Ambient.r = (float)(x / 256);
	mtrl.Diffuse.g = mtrl.Ambient.g = (float)(y / 256);
	mtrl.Diffuse.b = mtrl.Ambient.b = (float)(z / 256);
	mtrl.Diffuse.a = mtrl.Ambient.a = 1.0f;
	return mtrl;
}

__device__ 
sp::sp(const D3DMATERIAL9& mtrl)
{
	x = 256 * mtrl.Diffuse.r;
	y = 256 * mtrl.Diffuse.g;
	z = 256 * mtrl.Diffuse.b;
}

__device__
void sp::getPOINT(long& ox, long& oy, long cx, long cy) const
{
	if (z > -20) {
		ox = (long)(((x / (PERSPECTIVE_RATIO * (z + 20))) + 10) * cx / 20);
		oy = (long)(((y / (PERSPECTIVE_RATIO * (z + 20))) + 10) * cx / 20);
	}
	else {
		ox = (long)(((x / 1E-10) + 10) * cx / 20);
		oy = (long)(((y / 1E-10) + 10) * cx / 20);
	}
}
