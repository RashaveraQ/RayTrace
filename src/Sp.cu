#include "hip/hip_runtime.h"
#pragma once

#include <math.h>
#include "sp.cuh"
#include "Matrix.cuh"

__device__
Sp::Sp(const Matrix& m)
{
	x = m.get_data(1,1);
	y = m.get_data(2,1);
	z = m.get_data(3,1);
}

__device__
bool Sp::operator==( const Sp& a ) const
{
	return x == a.x && y == a.y && z == a.z;
}

__device__ 
bool Sp::operator!=(const Sp& a) const
{
	return x != a.x || y != a.y || z != a.z;
}

__device__ 
Sp&	Sp::operator=(const Sp& a)
{
	x = a.x;
	y = a.y;
	z = a.z;
	return *this;
}

__device__ 
float	Sp::operator*(const Sp& a) const
{
	return x*a.x+y*a.y+z*a.z;
}

__device__ 
Sp Sp::operator+(const Sp& a) const
{
	return Sp( x+a.x, y+a.y, z+a.z );
}

__device__
Sp Sp::operator-(const Sp& a) const
{
	return Sp( x-a.x, y-a.y, z-a.z );
}

__device__
Sp Sp::operator-( void ) const
{
	return Sp( -x, -y, -z );
}

__device__
Sp Sp::operator*(float k) const
{
	return Sp( x*k, y*k, z*k );
}

__device__
Sp Sp::operator/(float a) const
{
	return Sp( x/a, y/a, z/a );
}
/*
void Sp::print()
{
	printf( "(%f,%f,%f)\n", x, y, z );
}
*/

__device__ 
Sp	operator*(float k, const Sp& a)
{
	return a*k;
}

__device__
Sp	Sp::e() const
{
	float	r = sqrt(x*x + y*y + z*z);

	return Sp( x/r, y/r, z/r );
}

/*
POINT sp::getPOINT(const CSize& size) const
{
	POINT ans;
	if (z > -20) {
		ans.x = (long)(((x / (PERSPECTIVE_RATIO * (z + 20))) + 10) * size.cx / 20);
		ans.y = (long)(((y / (PERSPECTIVE_RATIO * (z + 20))) + 10) * size.cx / 20);
	} else {
		ans.x = (long)(((x / 1E-10) + 10) * size.cx / 20);
		ans.y = (long)(((y / 1E-10) + 10) * size.cx / 20);
	}
	return ans;
}
*/

__device__
D3DMATERIAL9 Sp::getMaterial() const
{
	D3DMATERIAL9	mtrl;
	ZeroMemory(&mtrl, sizeof(D3DMATERIAL9));
	mtrl.Diffuse.r = mtrl.Ambient.r = (float)(x / 256);
	mtrl.Diffuse.g = mtrl.Ambient.g = (float)(y / 256);
	mtrl.Diffuse.b = mtrl.Ambient.b = (float)(z / 256);
	mtrl.Diffuse.a = mtrl.Ambient.a = 1.0f;
	return mtrl;
}

__device__ 
Sp::Sp(const D3DMATERIAL9& mtrl)
{
	x = 256 * mtrl.Diffuse.r;
	y = 256 * mtrl.Diffuse.g;
	z = 256 * mtrl.Diffuse.b;
}
