#include "hip/hip_runtime.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "NurbsPlane.cuh"

__device__
DevNurbsPlane::DevNurbsPlane(DevNode** const root, const sp Color)
	: DevNode(root, PLANE, Color)
{

}

__device__
bool DevNurbsPlane::IsInside(const sp& L) const
{
	return (L.y >= 0.0);
}

__device__
bool DevNurbsPlane::GetInfo(const sp& K, const sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide) const
{
	if (pHint && pHint->pNode == this && fromOutSide)
		return false;

	float	t = (K.y) ? -L.y / K.y : ((L.y > 0) ? FLT_MAX : -FLT_MAX);

	if (t <= 0)
		return false;

	info.Cross = K * t + L;
	info.Vertical = sp(0, 1, 0);
	info.Distance = t * sqrt(K * K);
	info.isEnter = (L.y < 0);
	info.Material = GetPixel(info.Cross.x, info.Cross.z).getMaterial();
	info.pNode = this;

	return -3 < info.Cross.x && info.Cross.x < 3 && -3 < info.Cross.z && info.Cross.z < 3;
}

__global__
void newNurbsPlane(DevNode** out, DevNode** const root, const D3DMATERIAL9 Material)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		*out = new DevNurbsPlane(root, sp(Material));
}

bool newDevNurbsPlane(DevNode*** out, DevNode** const root, const D3DMATERIAL9 Material)
{
	if (!mallocDev(out))
		return false;

	newNurbsPlane<<<1, 1>>>(*out, root, Material);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}