#include "hip/hip_runtime.h"
﻿#include <math.h>
#include "Node.cuh"
#include "Info.cuh"
#include "DoCuda.h"

__device__
DevNode::DevNode(DevNode** const root, node_type NodeType, const Sp Color)
	: m_Root(root), m_pParent(0), m_NodeType(NodeType), m_Reflect(0), m_Through(0), m_Refractive(1)
{
	m_Material = Color.getMaterial();
}

__device__
DevNode::DevNode(const DevNode& other) : m_Root(other.m_Root), m_Scale(4, 4), m_Rotate(4, 4), m_Move(4, 4), m_Matrix(4, 4)
{
	m_NodeType = other.m_NodeType;
	m_Material = other.m_Material;
	m_Scale = other.m_Scale;
	m_Rotate = other.m_Rotate;
	m_Move = other.m_Move;
	m_Matrix = other.m_Matrix;
	m_Reflect = other.m_Reflect;
	m_Through = other.m_Through;
	m_Refractive = other.m_Refractive;
}

__device__
Sp DevNode::GetColor(const Sp& K, const Sp& L, int nest, const DevInfo* pHint, bool fromOutSide)
{
	DevInfo	info;

	// 再帰数が１０を越える又は、交点が存在しない場合、
	if (nest > 10 || !GetInfo2(K, L, info, pHint, fromOutSide))
		return Sp(127, 127, 127);

	Sp k = K.e();
	Sp v = info.Vertical.e();

	Sp k2 = k - 2 * (v * k) * v;
	Sp l2 = info.Cross;


	// 反射率がある場合、
	if (info.pNode->m_Reflect > 0) {
		// 反射した視線ベクトルから色を取得。
		Sp c = (*m_Root)->GetColor(k2, l2, nest + 1, &info, true);
		// 反射率で色を混ぜる。
		info.Material = (info.pNode->m_Reflect * c + (1 - info.pNode->m_Reflect) * Sp(info.Material)).getMaterial();
	}


	// 透過率がある場合、
	if (info.pNode->m_Through > 0) {
		float r = info.Refractive;
		float i = k * v;
		// 全反射でない場合、
		if (r > 1.0 || asin(r) > acos(-i)) {
			k2 = r * (k + v) - v;
			l2 = info.Cross;
			fromOutSide = !fromOutSide;
		}
		// 屈折した視線ベクトルから色を取得。
		Sp c = (*m_Root)->GetColor(k2, l2, nest + 1, &info, fromOutSide);
		// 透過率で色を混ぜる。
		info.Material = (info.pNode->m_Through * c + (1 - info.pNode->m_Through) * Sp(info.Material)).getMaterial();
	}

	Sp Light = Sp(1, 1, 1);
	// 光源より色を補正。
	float	x = -Light.e() * info.Vertical.e();
	x = (x > 0.0) ? x : 0.0;
	float t = 64 + 191 * sin(M_PI / 2 * x);
	float b = 191 * (1 - cos(M_PI / 2 * x));

	return (t - b) * Sp(info.Material) / 255 + Sp(b, b, b);
}

// 視線ベクトル(Kt+L)と交差する物体の情報infoを返す。
// 戻り値:true 交差あり,false 交差なし
__device__
bool DevNode::GetInfo2(const Sp& K, const Sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide)
{
	// START Boundary 
	/*
	float a = gK * gK;
	float b = (gL - m_Boundary.Center) * gK;
	float c = (m_Boundary.Center - gL) * (m_Boundary.Center - gL) - m_Boundary.Radius * m_Boundary.Radius;
	float bb_ac = b*b-a*c;

	if (bb_ac < 0)
	return FALSE;

	float t1, t2;

	t1 = (-b+sqrt(bb_ac))/a;
	t2 = (-b-sqrt(bb_ac))/a;

	if (t1 <= 0 || t2 <= 0)
	return FALSE;
	// End Boundary
	*/

	Matrix& m = m_Matrix;
	const Matrix& Inv_m = m.Inv();

	Sp L2 = Inv_m * L;
	Sp K2 = Inv_m * (K + L) - L2;

	if (!GetInfo(K2, L2, info, pHint, fromOutSide)) {
		return false;
	}

	info.Vertical = m_Scale.Inv() * info.Vertical;
	info.Vertical = m * (info.Vertical + info.Cross) - m * info.Cross;
	info.Cross = m * info.Cross;
	info.Distance = (info.Cross - L).abs();
	info.Refractive = info.pNode->m_Refractive / ((pHint) ? pHint->Refractive : 1.0);
	if (!info.isEnter)
		info.Refractive = 1 / info.Refractive;

	return true;
}

__device__
Sp DevNode::GetPixel(float x, float y) const
{
//	COLORREF	c;

//	if (m_TextureFileName.IsEmpty())
		return Sp(256 * m_Material.Diffuse.r, 256 * m_Material.Diffuse.g, 256 * m_Material.Diffuse.b);

	//c = m_TextureDC.GetPixel((int)(x * m_TextureSize.cx), (int)(y * m_TextureSize.cy));

	//if (c == -1)
	//	return Sp(256 * m_Material.Diffuse.r, 256 * m_Material.Diffuse.g, 256 * m_Material.Diffuse.b);

	//return Sp(GetRValue(c), GetGValue(c), GetBValue(c));
}

__device__
bool DevNode::IsInside2(const Sp& L) {
	return IsInside(m_Matrix.Inv() * L);
}