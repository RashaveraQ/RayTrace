#include "hip/hip_runtime.h"
﻿#include <math.h>
#include "Node.cuh"
#include "Info.cuh"
#include "DoCuda.h"

__device__
DevNode::DevNode(DevNode** const root, node_type NodeType, const sp Color)
	: m_Root(root), m_pParent(0), m_NodeType(NodeType), m_Material(Color.getMaterial()), m_Reflect(0), m_Through(0), m_Refractive(1)
{
}

__device__
DevNode::DevNode(const DevNode& other)
	: m_Root(other.m_Root), m_Scale(other.m_Scale), m_Rotate(other.m_Rotate), m_Move(other.m_Move), m_Matrix(other.m_Matrix),
	m_NodeType(other.m_NodeType), m_Material(other.m_Material), m_Reflect(other.m_Reflect), m_Through(other.m_Through), m_Refractive(other.m_Refractive)
{
}

__device__
sp DevNode::GetColor(const sp& K, const sp& L, int nest, const DevInfo* pHint, bool fromOutSide)
{
	DevInfo	info;

	// 再帰数が１０を越える又は、交点が存在しない場合、
	if (nest > 10 || !GetInfo2(K, L, info, pHint, fromOutSide))
		return sp(127, 127, 127);

	sp k = K.e();
	sp v = info.Vertical.e();

	sp k2 = k - 2 * (v * k) * v;
	sp l2 = info.Cross;


	// 反射率がある場合、
	if (info.pNode->m_Reflect > 0) {
		// 反射した視線ベクトルから色を取得。
		sp c = (*m_Root)->GetColor(k2, l2, nest + 1, &info, true);
		// 反射率で色を混ぜる。
		info.Material = (info.pNode->m_Reflect * c + (1 - info.pNode->m_Reflect) * sp(info.Material)).getMaterial();
	}


	// 透過率がある場合、
	if (info.pNode->m_Through > 0) {
		float r = info.Refractive;
		float i = k * v;
		// 全反射でない場合、
		if (r > 1.0 || asin(r) > acos(-i)) {
			k2 = r * (k + v) - v;
			l2 = info.Cross;
			fromOutSide = !fromOutSide;
		}
		// 屈折した視線ベクトルから色を取得。
		sp c = (*m_Root)->GetColor(k2, l2, nest + 1, &info, fromOutSide);
		// 透過率で色を混ぜる。
		info.Material = (info.pNode->m_Through * c + (1 - info.pNode->m_Through) * sp(info.Material)).getMaterial();
	}

	sp Light = sp(1, 1, 1);
	// 光源より色を補正。
	float	x = -Light.e() * info.Vertical.e();
	x = (x > 0.0) ? x : 0.0;
	float t = 64 + 191 * sin(M_PI / 2 * x);
	float b = 191 * (1 - cos(M_PI / 2 * x));

	return (t - b) * sp(info.Material) / 255 + sp(b, b, b);
}

// 視線ベクトル(Kt+L)と交差する物体の情報infoを返す。
// 戻り値:true 交差あり,false 交差なし
__device__
bool DevNode::GetInfo2(const sp& K, const sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide)
{
	// START Boundary 
	/*
	float a = gK * gK;
	float b = (gL - m_Boundary.Center) * gK;
	float c = (m_Boundary.Center - gL) * (m_Boundary.Center - gL) - m_Boundary.Radius * m_Boundary.Radius;
	float bb_ac = b*b-a*c;

	if (bb_ac < 0)
	return FALSE;

	float t1, t2;

	t1 = (-b+sqrt(bb_ac))/a;
	t2 = (-b-sqrt(bb_ac))/a;

	if (t1 <= 0 || t2 <= 0)
	return FALSE;
	// End Boundary
	*/

	matrix& m = m_Matrix;
	const matrix& Inv_m = m.Inv();

	sp L2 = Inv_m * L;
	sp K2 = Inv_m * (K + L) - L2;

	if (!GetInfo(K2, L2, info, pHint, fromOutSide)) {
		return false;
	}

	info.Vertical = m_Scale.Inv() * info.Vertical;
	info.Vertical = m * (info.Vertical + info.Cross) - m * info.Cross;
	info.Cross = m * info.Cross;
	info.Distance = (info.Cross - L).abs();
	info.Refractive = info.pNode->m_Refractive / ((pHint) ? pHint->Refractive : 1.0);
	if (!info.isEnter)
		info.Refractive = 1 / info.Refractive;

	return true;
}

__device__
sp DevNode::GetPixel(float x, float y) const
{
//	COLORREF	c;

//	if (m_TextureFileName.IsEmpty())
		return sp(256 * m_Material.Diffuse.r, 256 * m_Material.Diffuse.g, 256 * m_Material.Diffuse.b);

	//c = m_TextureDC.GetPixel((int)(x * m_TextureSize.cx), (int)(y * m_TextureSize.cy));

	//if (c == -1)
	//	return sp(256 * m_Material.Diffuse.r, 256 * m_Material.Diffuse.g, 256 * m_Material.Diffuse.b);

	//return sp(GetRValue(c), GetGValue(c), GetBValue(c));
}

__device__
bool DevNode::IsInside2(const sp& L) {
	return IsInside(m_Matrix.Inv() * L);
}