#include "hip/hip_runtime.h"
#include <math.h>
#include "Node.cuh"
#include "Info.cuh"

__device__
Sp DevNode::GetColor(const Sp& K, const Sp& L, int nest, const DevInfo* pHint, bool fromOutSide)
{
	DevInfo	info;

	// �ċA�����P�O���z���閔�́A��_�����݂��Ȃ��ꍇ�A
	if (nest > 10 || !GetInfo2(K, L, info, pHint, fromOutSide))
		return Sp(127, 127, 127);

	Sp k = K.e();
	Sp v = info.Vertical.e();

	Sp k2 = k - 2 * (v * k) * v;
	Sp l2 = info.Cross;


	// ���˗�������ꍇ�A
	if (info.pNode->m_Reflect > 0) {
		// ���˂��������x�N�g������F���擾�B
		Sp c = m_Root->GetColor(k2, l2, nest + 1, &info, true);
		// ���˗��ŐF��������B
		info.Material = (info.pNode->m_Reflect * c + (1 - info.pNode->m_Reflect) * Sp(info.Material)).getMaterial();
	}


	// ���ߗ�������ꍇ�A
	if (info.pNode->m_Through > 0) {
		double r = info.Refractive;
		double i = k * v;
		// �S���˂łȂ��ꍇ�A
		if (r > 1.0 || asin(r) > acos(-i)) {
			k2 = r * (k + v) - v;
			l2 = info.Cross;
			fromOutSide = !fromOutSide;
		}
		// ���܂��������x�N�g������F���擾�B
		Sp c = m_Root->GetColor(k2, l2, nest + 1, &info, fromOutSide);
		// ���ߗ��ŐF��������B
		info.Material = (info.pNode->m_Through * c + (1 - info.pNode->m_Through) * Sp(info.Material)).getMaterial();
	}

	Sp Light = Sp(1, 1, 1);
	// �������F��␳�B
	double	x = -Light.e() * info.Vertical.e();
	x = (x > 0.0) ? x : 0.0;
	double t = 64 + 191 * sin(M_PI / 2 * x);
	double b = 191 * (1 - cos(M_PI / 2 * x));

	return (t - b) * Sp(info.Material) / 255 + Sp(b, b, b);
}

// �����x�N�g��(Kt+L)�ƌ������镨�̂̏��info��Ԃ��B
// �߂�l:true ��������,false �����Ȃ�
__device__
bool DevNode::GetInfo2(const Sp& K, const Sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide)
{
	// START Boundary 
	/*
	double a = gK * gK;
	double b = (gL - m_Boundary.Center) * gK;
	double c = (m_Boundary.Center - gL) * (m_Boundary.Center - gL) - m_Boundary.Radius * m_Boundary.Radius;
	double bb_ac = b*b-a*c;

	if (bb_ac < 0)
	return FALSE;

	double t1, t2;

	t1 = (-b+sqrt(bb_ac))/a;
	t2 = (-b-sqrt(bb_ac))/a;

	if (t1 <= 0 || t2 <= 0)
	return FALSE;
	// End Boundary
	*/
	
	Matrix m = m_Matrix;
	Matrix Inv_m = m.Inv();

	Sp L2 = Inv_m * L;
	Sp K2 = Inv_m * (K + L) - L2;

	if (!GetInfo(K2, L2, info, pHint, fromOutSide)) {
		return false;
	}

	info.Vertical = m_Scale.Inv() * info.Vertical;
	info.Vertical = m * (info.Vertical + info.Cross) - m * info.Cross;
	info.Cross = m * info.Cross;
	info.Distance = (info.Cross - L).abs();
	info.Refractive = info.pNode->m_Refractive / ((pHint) ? pHint->Refractive : 1.0);
	if (!info.isEnter)
		info.Refractive = 1 / info.Refractive;

	return true;
}
