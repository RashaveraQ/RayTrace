#include "hip/hip_runtime.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "Torus.cuh"
#include "Solve_Polynomial.cuh"

__device__
DevTorus::DevTorus(DevNode** const root, const char* const Name, const Sp Color)
	: DevNode(root, TORUS, Name, Color), m_R(0.7f), m_r(0.3f)
{

}

__device__
bool DevTorus::IsInside(const Sp& L) const
{
	float	d;

	d = m_R - sqrt(L.x * L.x + L.y * L.y);
	d *= d;

	return (m_r * m_r <= d + L.z * L.z);
}

__device__
bool DevTorus::GetInfo(const Sp& K, const Sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide) const
{
	float	R2 = m_R * m_R;
	float	a = K * K;
	float	b = 2 * (K * L);
	float	c = L * L + R2 - m_r * m_r;

	float	k[20], r[50];
	int		n;

	k[4] = a * a;
	k[3] = 2 * a * b;
	k[2] = 2 * a * c + b * b - 4 * R2 * (K.x * K.x + K.y * K.y);
	k[1] = 2 * b * c - 8 * R2 * (K.x * L.x + K.y * L.y);
	k[0] = c * c - 4 * R2 * (L.x * L.x + L.y * L.y);

	n = Solve_Polynomial(4, k, 0, 20000, r);

	if (n == 0)
		return FALSE;

	if (pHint && pHint->pNode == this) {
		if (fromOutSide && n % 2 == 1) {
			if (n == 1)
				return false;
			r[0] = r[1];
		}
		if (!fromOutSide && n % 2 == 0) {
			r[0] = r[1];
		}
	}

	Sp		p;
	float	th;

	info.isEnter = IsInside(L) == TRUE ? 1 : 0;
	info.Cross = p = K * r[0] + L;

	if (p.x == 0.0) {
		info.Vertical = Sp(0, p.y - ((p.y > 0) ? 1 : -1) * m_R, p.z);
	}
	else {
		th = atan2(p.y, p.x);
		info.Vertical = Sp(p.x - m_R * cos(th), p.y - m_R * sin(th), p.z);
	}

	info.Distance = r[0] * sqrt(K * K);

	float x, y, z, phy;

	x = info.Vertical.x;
	y = info.Vertical.y;
	z = info.Vertical.z;

	th = acosf(y) / (2 * M_PI); if (x < 0) th = 1 - th;
	phy = acosf(z / sqrtf(x * x + z * z)) / (2 * M_PI);
	if (x < 0)
		phy = 1 - phy;

	info.Material = GetPixel(phy, th).getMaterial();
	info.pNode = this;

	return true;
}

__global__
void newTorus(DevNode** out, DevNode** const root, const char* const Name, const D3DMATERIAL9 Material)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		*out = new DevTorus(root, Name, Sp(Material));
}

bool newDevTorus(DevNode*** out, DevNode** const root, const char* const Name, const D3DMATERIAL9 Material)
{
	if (!mallocDev(out))
		return false;

	newTorus<<<1, 1>>>(*out, root, Name, Material);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}