﻿#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <math.h>

#define ENOUGH_SMALL 0.00001f

__host__ __device__
static float func(int d, float *k, float x)
{
	int		i;
	float	ans = 0;

	for (i = 0; i <= d; i++) ans += k[i] * pow(x, i);

	return ans;
}

__host__ __device__
static int compare(const void *arg1, const void *arg2)
{
	float *p1, *p2;

	p1 = (float*)arg1;
	p2 = (float*)arg2;

	if (*p1 == *p2) return 0;
	else if (*p1 > *p2) return 1;
	else return -1;
}

__host__ __device__
void qsort2(float* base, size_t num, size_t width, int(*compare)(const void *, const void *))
{
	if (num < 2)
		return;

	for (unsigned int i = 0; i < num - 1; i++) {
		for (unsigned int j = i; j < num - 1; j++) {
			if (compare(base+ j, base + j + 1) > 0) {
				float v = base[j];
				base[j] = base[j+1];
				base[j+1] = v;
			}
		}
	}
}

__host__ __device__
static float walk(int d, float *k, float min, float max)
{
	float a, b, f, g, x, y;

	long l = 0;

	if ((f = func(d, k, max)) == 0.0f) return max;
	if ((g = func(d, k, min)) == 0.0f) return min;
	
	if (f * g > 0) {
		//MessageBox ( NULL, "Error!", "Solve_Polynomial", MB_OK );
		//exit(0);
	}
	
	a = min, b = max;
	while (1) {
		if (fabs(a - b) < ENOUGH_SMALL)
			break;

		x = (a + b) / 2;	// CPU error!?
		y = func(d, k, x);

		if (f > 0) {
			if (func(d, k, b) - y < ENOUGH_SMALL)
				break;

			if (y < 0)
				a = x;
			else
				b = x;
		}
		else {
			if (func(d, k, a) - y < ENOUGH_SMALL)
				break;

			if (y > 0)
				a = x;
			else
				b = x;
		}

		if (l++ > 100)
			break;			// for CPU bug
	}
	return x;
}

__host__ __device__
int Solve_Polynomial(int d, float *k, float min, float max, float *r)
{
	int		i, j, N, ans;
	float	D;
	float	*K, *R, *y;

	switch (d)
	{
	case 1:
		if (k[1] == 0.0f)
			return 0;

		r[0] = -k[0] / k[1];
		return (min <= r[0] && r[0] <= max) ? 1 : 0;

	case 2:
		if (k[2] == 0.0f)
		{
			if (k[1] == 0.0f) return 0;

			r[0] = -k[0] / k[1];
			return (min <= r[0] && r[0] <= max) ? 1 : 0;
		}

		D = k[1] * k[1] - 4 * k[2] * k[0];

		if (D == 0.0f)
		{
			r[0] = -k[1] / (2.0f * k[2]);
			return (min <= r[0] && r[0] <= max) ? 1 : 0;
		}
		else if (D > 0)
		{
			r[0] = (-k[1] + sqrt(D)) / (2.0f * k[2]);
			r[1] = (-k[1] - sqrt(D)) / (2.0f * k[2]);
			if (r[0] > r[1])
				D = r[0], r[0] = r[1], r[1] = D;
			i = ((min <= r[0] && r[0] <= max) ? 1 : 0) + ((min <= r[1] && r[1] <= max) ? 1 : 0);
			return i;
		}
		return 0;
	}

	K = new float[2 * d];
	R = new float[2 * d + 4];
	y = new float[2 * d + 4];

	for (i = 1; i <= d; i++)
		K[i - 1] = i * k[i] / k[d] / d;

	// 極値の個数を求める。
	N = Solve_Polynomial(d - 1, K, min, max, R + 1);
	R[0] = min, R[N + 1] = max;

	for (i = 0; i <= N + 1; i++)
		y[i] = func(d, k, R[i]);

	for (ans = i = 0; i <= N; i++)
	{
		// 解が存在しない区間の場合、次へ。
		if (y[i] * y[i + 1] > 0) continue;

		r[ans++] = walk(d, k, R[i], R[i + 1]);
	}

	qsort2(r, (size_t)ans, sizeof(float), compare);

	for (i = 0; i < ans - 1; i++)
	{
		if (r[i + 1] - r[i] <= ENOUGH_SMALL)
		{
			for (j = i; j < ans - 1; j++)
				r[j] = r[j + 1];
			i--;
			ans--;
		}
	}

	delete K;
	delete R;
	delete y;

	return ans;
}

