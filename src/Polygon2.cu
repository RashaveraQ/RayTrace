#include "hip/hip_runtime.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "Polygon2.cuh"

__device__
DevPolygon2::DevPolygon2(const DevPolygon2& other) : DevNode(other)
{
	int	i;

	m_N = other.m_N;
	m_P = new sp[m_N];
	for (i = 0; i < m_N; i++)
		m_P[i] = other.m_P[i];
}

__device__
bool DevPolygon2::IsInside(const sp& L) const
{
	if (L.z < 0.0)
		return false;

	bool	r = false;
	int		i, j;

	for (i = 0, j = m_N - 1; i < m_N; j = i++)
	{
		if ((
			((m_P[i].y <= L.y) && (L.y < m_P[j].y))
			||
			((m_P[j].y <= L.y) && (L.y < m_P[i].y))
			)
			&&
			(
			L.x < (m_P[j].x - m_P[i].x) * (L.y - m_P[i].y) / (m_P[j].y - m_P[i].y) + m_P[i].x
			)
			)
			r = !r;
	}
	return r;
}

__device__
bool DevPolygon2::GetInfo(const sp& K, const sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide) const
{
	float	a = K * K;
	float	b = K * L;
	float	c = L * L - 1.0f;

	float	bb_ac = b*b - a*c;

	if (bb_ac < 0)
		return false;

	float	t;
	float	t1 = (-b + sqrt(bb_ac)) / a;
	float	t2 = (-b - sqrt(bb_ac)) / a;

	if (t1 > 0)
	{
		if (t2 > 0)
		{
			t = (t1 < t2) ? t1 : t2;
		}
		else
			t = t1;
	}
	else
	{
		if (t2 > 0)
			t = t2;
		else
			return false;
	}

	info.Cross = info.Vertical = K * t + L;
	info.Distance = t * sqrt(K * K);
	info.Material = m_Material;
	info.pNode = this;

	return true;
}

__global__
void newPolygon2(DevNode** out, DevNode** const root, const D3DMATERIAL9 Material)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		*out = new DevPolygon2(root, sp(Material));
}

bool newDevPolygon2(DevNode*** out, DevNode** const root, const D3DMATERIAL9 Material)
{
	if (!mallocDev(out))
		return false;

	newPolygon2<<<1, 1>>>(*out, root, Material);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}