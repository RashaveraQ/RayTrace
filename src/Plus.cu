#include "hip/hip_runtime.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "Plus.cuh"

__device__
bool DevPlus::IsInside(const Sp& L) const
{
	for (int i = 0; i < m_Member; i++) {
		if (m_Node[i]->IsInside2(L))
			return TRUE;
	}

	return FALSE;
}

__device__
bool DevPlus::GetInfo(const Sp& K, const Sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide) const
{
	DevInfo	tmp;
	//int		n;
	float	l = -1;

	for (int i = 0; i < m_Member; i++) {
		if (m_Node[i]->GetInfo2(K, L, tmp, pHint, fromOutSide)) {
			if (l == -1 || tmp.Distance < l) {
				l = tmp.Distance;
				//n = i;
				info = tmp;
			}
		}
	}

	if (l < 0)
		return false;

	if (info.Material.Diffuse.r < 0)
		info.Material = m_Material;

	return true;
}

__global__
void newPlus(DevNode** out, DevNode** const root, const char* const Name)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		*out = new DevPlus(*root, Name);
}

bool newDevPlus(DevNode** out, DevNode** const root, const char* const Name)
{
	if (!mallocDev(out))
		return false;

	newPlus<<<1, 1>>>(out, root, Name);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}