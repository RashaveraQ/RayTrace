#include "hip/hip_runtime.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "NurbsPrimitive.cuh"

DevNurbsPrimitive::DevNurbsPrimitive(DevNode** const root, const sp Color)
	: DevNode(root, ePLANE, Color)
{

}

bool DevNurbsPrimitive::IsInside(const sp& L) const
{
	return (L.y >= 0.0);
}

bool DevNurbsPrimitive::GetInfo(const sp& K, const sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide) const
{
	if (pHint && pHint->pNode == this && fromOutSide)
		return false;

	float	t = (K.y) ? -L.y / K.y : ((L.y > 0) ? FLT_MAX : -FLT_MAX);

	if (t <= 0)
		return false;

	info.Cross = K * t + L;
	info.Vertical = sp(0, 1, 0);
	info.Distance = t * sqrt(K * K);
	info.Material = GetPixel(info.Cross.x, info.Cross.z).getMaterial();
	info.pNode = this;

	return -3 < info.Cross.x && info.Cross.x < 3 && -3 < info.Cross.z && info.Cross.z < 3;
}

__global__
void newNurbsPrimitive(DevNode** out, DevNode** const root, const D3DMATERIAL9 Material)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		*out = new DevNurbsPrimitive(root, sp(Material));
}

bool newDevNurbsPrimitive(DevNode*** out, DevNode** const root, const D3DMATERIAL9 Material)
{
	if (!mallocDev(out))
		return false;

	newNurbsPrimitive<<<1, 1>>>(*out, root, Material);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}