#include "hip/hip_runtime.h"
﻿#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "Multiple.cuh"

__device__
bool DevMultiple::IsInside(const Sp& L) const
{
	if (!m_Member)
		return false;

	for (int i = 0; i < m_Member; i++) {
		if (!m_Node[i]->IsInside2(L))
			return false;
	}
	return true;
}

__device__
bool DevMultiple::GetInfo(const Sp& K, const Sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide) const
{
	if (!m_Member)
		return false;

	// すべての要素について、
	int	i;
	for (i = 0; i < m_Member; i++) {
		// 視点の先に、交点がない場合。
		if (!m_Node[i]->GetInfo2(K, L, info, pHint, fromOutSide))
			return false;

		int j;
		for (j = 0; j < m_Member; j++) {
			if (i == j)
				continue;
			if (!m_Node[j]->IsInside2(info.Cross))
				break;
		}

		if (j == m_Member)
			break;
	}

	if (i == m_Member)
		return false;

	if (info.Material.Diffuse.r < 0)
		info.Material = m_Material;

	return true;
}

__global__
void newMultiple(DevNode** out, DevNode** const root)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		*out = new DevMultiple(root);
}

bool newDevMultiple(DevNode*** out, DevNode** const root)
{
	if (!mallocDev(out))
		return false;

	newMultiple<<<1, 1>>>(*out, root);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}