#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "Gathering.cuh"

__device__
DevGathering::DevGathering(const DevGathering& other) : DevNode(other), m_Member(other.m_Member)
{
	for (int i = 0; i < m_Member; i++)
		m_Node[i] = (class DevNode*)other.m_Node[i]->MakeCopy();
}

__device__
DevGathering::~DevGathering()
{
	for (short i = 0; i < m_Member; i++)
		delete m_Node[i];
}

__device__
bool DevGathering::Delete(DevNode* pNode)
{
	short	i, j;

	for (i = 0; i < m_Member; i++) {
		if (m_Node[i] == pNode) {
			delete pNode;
			m_Member--;
			for (j = i; j < m_Member; j++)
				m_Node[j] = m_Node[j + 1];
			return true;
		}
		else if (m_Node[i]->Delete(pNode))
			return true;
	}
	return false;
}

/*
__device__
BOOL DevGathering::AddNode(CTreeCtrl& c, HTREEITEM hitem, DevNode* pNode)
{
	if (m_Member >= 1000)
		return FALSE;

	m_Node[m_Member++] = pNode;

	pNode->InsertItem(c, hitem);

	//pNode->updateMatrix(this);

	pNode->SetParent(this);

	return TRUE;
}
*/
__device__
bool DevGathering::AddNode(DevNode** node)
{
	if (m_Member >= 1000)
		return false;

	m_Node[m_Member++] = *node;

	(*node)->SetParent(this);

	return true;
}

__device__
void DevGathering::SetRoot(DevNode** const root)
{
	m_Root = root;

	for (int i = 0; i < m_Member; i++)
		m_Node[i]->SetRoot(root);
}
