#include "hip/hip_runtime.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "Sphere.cuh"

__device__
DevSphere::DevSphere(DevNode* const root, const char* const Name, const Sp Color)
	: DevNode(root, SPHERE, Name, Color)
{

}

__device__
bool DevSphere::GetInfo(const Sp& K, const Sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide) const
{
	if (pHint && pHint->pNode == this && fromOutSide)
		return FALSE;

	float	a = K * K;
	float	b = K * L;
	float	c = L * L - 1.0f;

	float	bb_ac = b*b - a*c;

	if (bb_ac < 0)
		return FALSE;

	float	t;
	float	t1 = (-b + sqrt(bb_ac)) / a;
	float	t2 = (-b - sqrt(bb_ac)) / a;

	info.isEnter = 0;
	if (t1 > 0) {
		if (t2 > 0) {
			if (pHint && pHint->pNode == this && !fromOutSide) {
				t = (t1 < t2) ? t2 : t1;
			}
			else {
				t = (t1 < t2) ? t1 : t2;
				info.isEnter = 1;
			}
		}
		else {
			t = t1;
		}
	}
	else {
		if (t2 > 0) {
			t = t2;
		}
		else
			return FALSE;
	}

	info.Cross = K * t + L;
	info.Vertical = info.isEnter ? info.Cross : -info.Cross;
	info.Distance = t * sqrt(K * K);

	float x, y, z, th, phy;

	x = info.Vertical.x;
	y = info.Vertical.y;
	z = info.Vertical.z;

	th = atan2f(y, sqrtf(x*x + z*z)) / M_PI + .5f;
	phy = atan2f(x, -z) / (2 * M_PI) + .5f;

	info.Material = GetPixel(phy, th).getMaterial();
	info.pNode = this;

	return TRUE;
}

__global__
void newSphere(DevNode** out)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		*out = new DevSphere();
}

bool newDevSphere(DevNode** out)
{
	if (!mallocDev(out))
		return false;

	newSphere<<<1, 1>>>(out);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}