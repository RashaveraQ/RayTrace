#include "hip/hip_runtime.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "Cone.cuh"

__device__
DevCone::DevCone(DevNode** const root, const sp Color)
	: DevNode(root, CONE, Color)
{

}

__device__
bool DevCone::IsInside(const sp& L) const
{
	return (0 <= L.y && L.y <= 1 && sqrt(L.x * L.x + L.z * L.z) <= L.y);
}

__device__
bool DevCone::GetInfo(const sp& K, const sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide) const
{
	if (pHint && pHint->pNode == this && fromOutSide)
		return FALSE;

	if (L.y > 1 && K.y >= 0)
		return FALSE;

	float t[2];
	sp     v[2];
	int i = 0;

	t[0] = (1 - L.y) / K.y;
	if (t[0] > 0) {
		sp p = K * t[0] + L;
		if (p.x * p.x + p.z * p.z <= 1) {
			v[i] = sp(0, 1, 0);
			info.Material = GetPixel(.5f * (p.x + 1), .5f * (p.z + 1)).getMaterial();
			info.pNode = this;
			info.Refractive = m_Refractive;
			i++;
		}
	}

	float	a, b, c, d, t1, t2;

	c = K.x * L.y - K.y * L.x, c *= c, d = c;
	c = K.z * L.y - K.y * L.z, c *= c, d += c;
	c = K.x * L.z - K.z * L.x, c *= c, d -= c;

	if (d >= 0) {
		d = sqrt(d);
		a = -(K.x * L.x + K.z * L.z - K.y * L.y);
		b = K.x * K.x + K.z * K.z - K.y * K.y;

		t1 = (a + d) / b;
		sp p = K * t1 + L;
		if (p.y < 0 || p.y > 1 || p.x * p.x + p.z * p.z > 1)
			t1 = -1;

		t2 = (a - d) / b;
		p = K * t2 + L;
		if (p.y < 0 || p.y > 1 || p.x * p.x + p.z * p.z > 1)
			t2 = -1;

		if (t1 > 0) {
			if (t2 > 0) {
				t[i] = (t1 < t2) ? t1 : t2;
				v[i] = K * t[i] + L;
				v[i].y *= -1;
				i++;
				t[i] = (t1 < t2) ? t2 : t1;
				v[i] = K * t[i] + L;
				v[i].y *= -1;
				i++;
			}
			else {
				t[i] = t1;
				v[i] = K * t[i] + L;
				v[i].y *= -1;
				i++;
			}
		}
		else {
			if (t2 > 0) {
				t[i] = t2;
				v[i] = K * t[i] + L;
				v[i].y *= -1;
				i++;
			}
		}
	}

	switch (i) {
	case 0:
		return FALSE;
	case 1:
		break;
	default:
		if (pHint && pHint->pNode == this) {
			if (t[0] < t[1]) {
				t[0] = t[1];
				v[0] = v[1];
			}
		}
		else if (t[0] > t[1]) {
			t[0] = t[1];
			v[0] = v[1];
		}
		break;
	}

	info.isEnter = (pHint && pHint->pNode == this) ? fromOutSide : !IsInside(L);
	info.Cross = K * t[0] + L;
	info.Vertical = info.isEnter ? v[0] : -v[0];
	info.Distance = t[0] * sqrt(K * K);
	info.Material = m_Material;
	info.pNode = this;

	return TRUE;
}

__global__
void newCone(DevNode** out, DevNode** const root, const D3DMATERIAL9 Material)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		*out = new DevCone(root, sp(Material));
}

bool newDevCone(DevNode*** out, DevNode** const root, const D3DMATERIAL9 Material)
{
	if (!mallocDev(out))
		return false;

	newCone<<<1, 1>>>(*out, root, Material);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}