#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"

#include <stdio.h>
/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
*/

int numSMs = -1;

__global__
void deletePoint(DevNode* out)
{
	if (threadIdx.x == 0)
		delete out;
}

DevNode* mallocDevicePointer()
{
	if (!DoCuda_Init())
		return 0;

	hipError_t cudaStatus;

	DevNode* devPtr;

	cudaStatus = hipMalloc(&devPtr, sizeof(void*));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	return devPtr;
Error:
	return 0;
}

bool freeDevicePointer(DevNode* pNode)
{
	deletePoint<<<1, 1 >>>(pNode);
	hipFree(pNode);
	return true;
}

bool DoCuda_Init()
{
	if (numSMs > 0)
		return true;

	hipDeviceProp_t deviceProp;
	if (hipSuccess != hipSetDevice(0))
		return false;

	if (hipSuccess != hipGetDeviceProperties(&deviceProp, 0))
		return false;

	numSMs = deviceProp.multiProcessorCount;

	return true;
}

bool DoCuda_OnSize(void** dst, const int imageW, const int imageH)
{
	return hipSuccess == hipMalloc(dst, imageW * imageH * sizeof(unsigned long));
}

__global__
void RayTrace(unsigned long* dst, const int imageW, const int imageH, DevNode* root, const int gridWidth, const int numBlocks, const fsize* pView, const matrix* pMatrix)
{
	// loop until all blocks completed
	for (unsigned int blockIndex = blockIdx.x; blockIndex < numBlocks; blockIndex += gridDim.x)
	{
		unsigned int blockX = blockIndex % gridWidth;
		unsigned int blockY = blockIndex / gridWidth;

		// process this block
		const int ix = blockDim.x * blockX + threadIdx.x;
		const int iy = blockDim.y * blockY + threadIdx.y;

		if ((ix < imageW) && (iy < imageH))
		{
			// Output the pixel
			int pixel = imageW * iy + ix;
			dst[pixel] = pixel;
		}
	}
}

// The dimensions of the thread block
#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16

// Increase the grid size by 1 if the image width or height does not divide evenly
// by the thread block dimensions
inline int iDivUp(int a, int b)
{
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
} // iDivUp

bool DoCuda_OnDraw(unsigned long* out, void* d_dst, class DevNode* root, const int imageW, const int imageH, const fsize* pView, const matrix* pMatrix)
{
	dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
	dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

	int numWorkerBlocks = numSMs;

	RayTrace<<<numWorkerBlocks, threads>>>((unsigned long*)d_dst, imageW, imageH, root, grid.x, grid.x * grid.y, pView, pMatrix);

	// Copy output vector from GPU buffer to host memory.
	return hipSuccess == hipMemcpy(out, d_dst, imageW * imageH * sizeof(unsigned long), hipMemcpyDeviceToHost);
}

bool DoCuda_Free(void* dst)
{
	return (dst) ? hipSuccess == hipFree(dst) : false;
}

