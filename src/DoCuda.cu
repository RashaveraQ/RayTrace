#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "GetVectorFromPoint.cuh"
#include "Sp.cuh"
#include "matrix.h"
#include "Matrix.cuh"
#include "Node.cuh"
#include "Plus.cuh"

#include <stdio.h>

int numSMs = -1;

__global__
void deletePoint(DevNode** out)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		delete *out;
}

bool mallocDev(DevNode*** out)
{
	if (!DoCuda_Init())
		return false;

	hipError_t cudaStatus = hipMalloc(out, sizeof(void*));
	if (cudaStatus != hipSuccess)
		return false;

	return true;
}

bool freeDevicePointer(DevNode** pNode)
{
	deletePoint<<<1, 1>>>(pNode);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	cudaStatus = hipFree(pNode);
	if (cudaStatus != hipSuccess)
		return false;

	return true;
}

bool DoCuda_Init()
{
	if (numSMs > 0)
		return true;

	hipError_t cuResult;
	hipCtx_t context;
	hipDevice_t cuDevice;

	hipDeviceProp_t deviceProp;
	if (hipSuccess != hipSetDevice(0))
		return false;

	if (hipSuccess != hipGetDeviceProperties(&deviceProp, 0))
		return false;

	numSMs = deviceProp.multiProcessorCount;

	// Get handle for device 0
	cuResult = hipDeviceGet(&cuDevice, 0);
	cuResult = hipDevicePrimaryCtxRetain(&context, cuDevice);
	size_t value = 1024 * 16;
	cuResult = hipDeviceSetLimit(hipLimitStackSize, value);

	return true;
}

bool DoCuda_OnSize(void** dst, const int imageW, const int imageH)
{
	if (!DoCuda_Init())
		return false;

	if (hipSuccess != hipMalloc(dst, imageW * imageH * sizeof(unsigned long)))
		return false;

	return true;
}

__global__
void RayTrace(unsigned long* dst, const int imageW, const int imageH, DevNode** root, const int gridWidth, const int numBlocks, const fsize* pView, const Matrix* pMatrix)
{
	// loop until all blocks completed
	for (unsigned int blockIndex = blockIdx.x; blockIndex < numBlocks; blockIndex += gridDim.x)
	{
		unsigned int blockX = blockIndex % gridWidth;
		unsigned int blockY = blockIndex / gridWidth;

		// process this block
		const int ix = blockDim.x * blockX + threadIdx.x;
		const int iy = blockDim.y * blockY + threadIdx.y;

		if ((ix < imageW) && (iy < imageH))
		{
			// Output the pixel
			int pixel = imageW * iy + ix;
			Sp k, l;
			GetVectorFromPoint(k, l, ix, iy, pView, imageW, imageH, pMatrix);
			Sp c = (*root)->GetColor(k, l, 0, NULL, true);
			dst[pixel] = RGB(c.x, c.y, c.z);
		}
	}
}

// The dimensions of the thread block
#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16

// Increase the grid size by 1 if the image width or height does not divide evenly
// by the thread block dimensions
inline int iDivUp(int a, int b)
{
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
} // iDivUp

bool DoCuda_OnDraw(unsigned long* out, void* d_dst, class DevNode** root, const int imageW, const int imageH, const fsize* pView, const matrix* pMatrix)
{
	dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
	dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

	int numWorkerBlocks = numSMs;

	Matrix m(pMatrix->get_width(), pMatrix->get_height());
	for (int w = 1; w <= m.get_width(); w++)
		for (int h = 1; h <= m.get_height(); h++)
			m.set_data(w, h, pMatrix->get_data(w, h));

	Matrix* dev_Matrix = 0;
	hipError_t cudaStatus;
	size_t szMatrix = sizeof(Matrix);

	cudaStatus = hipMalloc((void**)&dev_Matrix, szMatrix);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	cudaStatus = hipMemcpy(dev_Matrix, &m, szMatrix, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	fsize* dev_view = 0;
	size_t szFsize = sizeof(fsize);

	cudaStatus = hipMalloc((void**)&dev_view, szFsize);
	if (cudaStatus != hipSuccess) {
		return false;
	}
	cudaStatus = hipMemcpy(dev_view, pView, szFsize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	//size_t stackSize = checkStackSize();

	RayTrace<<<numWorkerBlocks, threads>>>((unsigned long*)d_dst, imageW, imageH, root, grid.x, grid.x * grid.y, dev_view, dev_Matrix);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(out, d_dst, imageW * imageH * sizeof(unsigned long), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	cudaStatus = hipFree(dev_Matrix);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	cudaStatus = hipFree(dev_view);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}

bool DoCuda_Free(void* dst)
{
	if (!DoCuda_Init())
		return false;

	if (!dst)
		return false;

	hipError_t cudaStatus = hipFree(dst);

	if (hipSuccess != cudaStatus)
		return false;

	return true;
}

__global__
void updateMatrix(DevNode** out, const Matrix* pMatrix)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		(*out)->m_Matrix = *pMatrix;
}

bool DoCuda_updateMatrix(DevNode** devNode, const struct matrix* pMatrix)
{
	if (!DoCuda_Init())
		return false;

	Matrix m(pMatrix->get_width(), pMatrix->get_height());
	for (int w = 1; w <= m.get_width(); w++)
		for (int h = 1; h <= m.get_height(); h++)
			m.set_data(w, h, pMatrix->get_data(w, h));

	Matrix* dev_Matrix = 0;
	hipError_t cudaStatus;
	size_t szMatrix = sizeof(Matrix);

	cudaStatus = hipMalloc((void**)&dev_Matrix, szMatrix);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	cudaStatus = hipMemcpy(dev_Matrix, &m, szMatrix, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	updateMatrix<<<1, 1>>>(devNode, dev_Matrix);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	cudaStatus = hipFree(dev_Matrix);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}

__global__
void SetRoot(DevNode** out, DevNode** root)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		(*out)->SetRoot(root);
}

bool DoCuda_SetRoot(DevNode** devNode, DevNode** devRoot)
{
	if (!DoCuda_Init())
		return false;

	SetRoot << <1, 1 >> >(devNode, devRoot);

	hipError_t cudaStatus;
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}
	return true;
}

__global__
void AddNode(DevGathering **out, DevNode** node)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		(*out)->AddNode(node);
}

bool DoCuda_AddNode(DevGathering** devGathering, DevNode** devNode)
{
	if (!DoCuda_Init())
		return false;

	AddNode<<<1, 1>>>(devGathering, devNode);

	hipError_t cudaStatus;
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}
	return true;
}