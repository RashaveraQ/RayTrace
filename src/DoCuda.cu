#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "GetVectorFromPoint.cuh"
#include "sp.cuh"
#include "matrix.cuh"
#include "Node.cuh"
#include "Plus.cuh"
#include <d3d9.h>
#include <stdio.h>

int numSMs = -1;

__global__
void deletePoint(DevNode** out)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		delete *out;
}

bool mallocDev(DevNode*** out)
{
	if (!DoCuda_Init())
		return false;

	hipError_t cudaStatus = hipMalloc(out, sizeof(void*));
	if (cudaStatus != hipSuccess)
		return false;

	return true;
}

bool freeDevicePointer(DevNode** pNode)
{
	deletePoint<<<1, 1>>>(pNode);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	cudaStatus = hipFree(pNode);
	if (cudaStatus != hipSuccess)
		return false;

	return true;
}

bool DoCuda_Init()
{
	if (numSMs > 0)
		return true;

	hipCtx_t context;
	hipDevice_t cuDevice;

	hipDeviceProp_t deviceProp;
	if (hipSuccess != hipSetDevice(0) ||
	    hipSuccess != hipGetDeviceProperties(&deviceProp, 0))
		return false;

	numSMs = deviceProp.multiProcessorCount;

	size_t value = 1024 * 16;

	// Get handle for device 0
	if (hipSuccess != hipDeviceGet(&cuDevice, 0) ||
	    hipSuccess != hipDevicePrimaryCtxRetain(&context, cuDevice) ||
	    hipSuccess != hipDeviceSetLimit(hipLimitStackSize, value))
		return false;

	return true;
}

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b)
{
	return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
	return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
	r = clamp(r, 0.0f, 255.0f);
	g = clamp(g, 0.0f, 255.0f);
	b = clamp(b, 0.0f, 255.0f);
	return (int(b) << 16) | (int(g) << 8) | int(r);
}

__global__
void RayTrace(unsigned int* dst, const int imageW, const int imageH, DevNode** root, const int gridWidth, const int numBlocks, const fsize* pView, const matrix* pMatrix)
{
	// loop until all blocks completed
	for (unsigned int blockIndex = blockIdx.x; blockIndex < numBlocks; blockIndex += gridDim.x)
	{
		unsigned int blockX = blockIndex % gridWidth;
		unsigned int blockY = blockIndex / gridWidth;

		// process this block
		const int ix = blockDim.x * blockX + threadIdx.x;
		const int iy = blockDim.y * blockY + threadIdx.y;

		if ((ix < imageW) && (iy < imageH))
		{
			// Output the pixel
			sp k, l;
			GetVectorFromPoint(k, l, ix, iy, pView, imageW, imageH, pMatrix);
			sp c = (*root)->GetColor(k, l, 0, NULL, true);
			int pixel = imageW * (imageH - iy - 1) + ix;
			dst[pixel] = rgbToInt(c.x, c.y, c.z);
		}
	}
}

// The dimensions of the thread block
#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16

// Increase the grid size by 1 if the image width or height does not divide evenly
// by the thread block dimensions
inline int iDivUp(int a, int b)
{
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
} // iDivUp

bool DoCuda_OnDraw(unsigned int* out, class DevNode** root, const int imageW, const int imageH, const fsize* pView, const matrix* pMatrix)
{
	dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
	dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

	int numWorkerBlocks = numSMs;

	matrix m(pMatrix->get_width(), pMatrix->get_height());
	for (int w = 1; w <= m.get_width(); w++)
		for (int h = 1; h <= m.get_height(); h++)
			m.set_data(w, h, pMatrix->get_data(w, h));

	matrix* dev_Matrix = 0;
	hipError_t cudaStatus;
	size_t szMatrix = sizeof(matrix);

	cudaStatus = hipMalloc((void**)&dev_Matrix, szMatrix);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	cudaStatus = hipMemcpy(dev_Matrix, &m, szMatrix, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	fsize* dev_view = 0;
	size_t szFsize = sizeof(fsize);

	cudaStatus = hipMalloc((void**)&dev_view, szFsize);
	if (cudaStatus != hipSuccess) {
		return false;
	}
	cudaStatus = hipMemcpy(dev_view, pView, szFsize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	//size_t stackSize = checkStackSize();

	RayTrace<<<numWorkerBlocks, threads>>>((unsigned int*)out, imageW, imageH, root, grid.x, grid.x * grid.y, dev_view, dev_Matrix);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	cudaStatus = hipFree(dev_Matrix);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	cudaStatus = hipFree(dev_view);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}

__global__
void updateMatrix(DevNode** out, const matrix* pMatrix)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		(*out)->m_Matrix = *pMatrix;
}

bool DoCuda_updateMatrix(DevNode** devNode, const struct matrix* pMatrix)
{
	if (!DoCuda_Init())
		return false;

	matrix m(pMatrix->get_width(), pMatrix->get_height());
	for (int w = 1; w <= m.get_width(); w++)
		for (int h = 1; h <= m.get_height(); h++)
			m.set_data(w, h, pMatrix->get_data(w, h));

	matrix* dev_Matrix = 0;
	hipError_t cudaStatus;
	size_t szMatrix = sizeof(matrix);

	cudaStatus = hipMalloc((void**)&dev_Matrix, szMatrix);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	cudaStatus = hipMemcpy(dev_Matrix, &m, szMatrix, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	updateMatrix<<<1, 1>>>(devNode, dev_Matrix);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	cudaStatus = hipFree(dev_Matrix);
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}

__global__
void updateColor(DevNode** out, float r, float g, float b)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
		(*out)->m_Material.Diffuse.r = (*out)->m_Material.Ambient.r = r;
		(*out)->m_Material.Diffuse.g = (*out)->m_Material.Ambient.g = g;
		(*out)->m_Material.Diffuse.b = (*out)->m_Material.Ambient.b = b;
	}
}

bool DoCuda_updateColor(DevNode** devNode, float r, float g, float b)
{
	if (!DoCuda_Init())
		return false;

	hipError_t cudaStatus;

	updateColor<<<1, 1>>>(devNode, r, g, b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}

__global__
void updateMaterial(DevNode** out, float reflect, float refractive, float through)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
		(*out)->m_Reflect = reflect;
		(*out)->m_Refractive = refractive;
		(*out)->m_Through = through;
	}
}

bool DoCuda_updateMaterial(DevNode** devNode, float reflect, float refractive, float through)
{
	if (!DoCuda_Init())
		return false;

	hipError_t cudaStatus;

	updateMaterial<<<1, 1>>>(devNode, reflect, refractive, through);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}

__global__
void SetRoot(DevNode** out, DevNode** root)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		(*out)->SetRoot(root);
}

bool DoCuda_SetRoot(DevNode** devNode, DevNode** devRoot)
{
	if (!DoCuda_Init())
		return false;

	SetRoot << <1, 1 >> >(devNode, devRoot);

	hipError_t cudaStatus;
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}
	return true;
}

__global__
void AddNode(DevGathering **out, DevNode** node)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		(*out)->AddNode(node);
}

bool DoCuda_AddNode(DevGathering** devGathering, DevNode** devNode)
{
	if (!DoCuda_Init())
		return false;

	AddNode<<<1, 1>>>(devGathering, devNode);

	hipError_t cudaStatus;
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}
	return true;
}