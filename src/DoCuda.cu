#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "GetVectorFromPoint.cuh"
#include "Sp.cuh"
#include "matrix.h"
#include "Matrix.cuh"
#include "Node.cuh"

#include <stdio.h>

int numSMs = -1;

__global__
void deletePoint(DevNode** out)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		delete *out;
}

bool mallocDev(DevNode** out)
{
	if (!DoCuda_Init())
		return false;

	hipError_t cudaStatus = hipMalloc((void**)&out, sizeof(void*));
	if (cudaStatus != hipSuccess)
		return false;

	return true;
}

bool freeDevicePointer(DevNode** pNode)
{
	deletePoint<<<1, 1>>>(pNode);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	cudaStatus = hipFree(pNode);
	if (cudaStatus != hipSuccess)
		return false;

	return true;
}

bool DoCuda_Init()
{
	if (numSMs > 0)
		return true;

	hipDeviceProp_t deviceProp;
	if (hipSuccess != hipSetDevice(0))
		return false;

	if (hipSuccess != hipGetDeviceProperties(&deviceProp, 0))
		return false;

	numSMs = deviceProp.multiProcessorCount;

	return true;
}

bool DoCuda_OnSize(void** dst, const int imageW, const int imageH)
{
	if (!DoCuda_Init())
		return false;

	if (hipSuccess != hipMalloc(dst, imageW * imageH * sizeof(unsigned long)))
		return false;

	return true;
}

__global__
void RayTrace(unsigned long* dst, const int imageW, const int imageH, DevNode** root, const int gridWidth, const int numBlocks, const fsize* pView, const Matrix* pMatrix)
{
	// loop until all blocks completed
	for (unsigned int blockIndex = blockIdx.x; blockIndex < numBlocks; blockIndex += gridDim.x)
	{
		unsigned int blockX = blockIndex % gridWidth;
		unsigned int blockY = blockIndex / gridWidth;

		// process this block
		const int ix = blockDim.x * blockX + threadIdx.x;
		const int iy = blockDim.y * blockY + threadIdx.y;

		if ((ix < imageW) && (iy < imageH))
		{
			// Output the pixel
			int pixel = imageW * iy + ix;
			Sp k, l;
			GetVectorFromPoint(k, l, ix, iy, pView, imageW, imageH, pMatrix);
			Sp c = (*root)->GetColor(k, l, 0, NULL, true);
			dst[pixel] = RGB(c.x, c.y, c.z);
		}
	}
}

// The dimensions of the thread block
#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16

// Increase the grid size by 1 if the image width or height does not divide evenly
// by the thread block dimensions
inline int iDivUp(int a, int b)
{
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
} // iDivUp

bool DoCuda_OnDraw(unsigned long* out, void* d_dst, class DevNode** root, const int imageW, const int imageH, const fsize* pView, const matrix* pMatrix)
{
	dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
	dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

	int numWorkerBlocks = numSMs;

	Matrix m(pMatrix->get_width(), pMatrix->get_height());
	for (int w = 1; w <= m.get_width(); w++)
		for (int h = 1; h <= m.get_height(); h++)
			m.set_data(w, h, pMatrix->get_data(w, h));

	RayTrace<<<numWorkerBlocks, threads>>>((unsigned long*)d_dst, imageW, imageH, root, grid.x, grid.x * grid.y, pView, &m);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	// Copy output vector from GPU buffer to host memory.
	return hipSuccess == hipMemcpy(out, d_dst, imageW * imageH * sizeof(unsigned long), hipMemcpyDeviceToHost);
}

bool DoCuda_Free(void* dst)
{
	/*
	if (!DoCuda_Init())
		return false;

	if (!dst)
		false;
	*/

	if (hipSuccess != hipFree(dst))
		return false;

	return true;
}

