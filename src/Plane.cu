#include "hip/hip_runtime.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "Plane.cuh"

__device__
DevPlane::DevPlane(DevNode* const root, const char* const Name, const Sp Color)
	: DevNode(root, PLANE, Name, Color)
{

}

__device__
bool DevPlane::IsInside(const Sp& L) const
{
	return (L.z >= 0.0);
}

__device__
bool DevPlane::GetInfo(const Sp& K, const Sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide) const
{
	if (pHint && pHint->pNode == this && fromOutSide)
		return false;

	float	t = (K.z) ? -L.z / K.z : ((L.z > 0) ? FLT_MAX : -FLT_MAX);

	if (t <= 0)
		return false;

	info.Cross = K * t + L;
	info.Vertical = Sp(0, 0, -1);
	info.Distance = t * sqrt(K * K);
	info.isEnter = (L.z < 0);
	info.Material = GetPixel(info.Cross.x, info.Cross.y).getMaterial();
	info.pNode = this;

	return true;
}

__global__
void newPlane(DevNode** out, DevNode** const root, const char* const Name, const D3DMATERIAL9 Material)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		*out = new DevPlane(*root, Name, Sp(Material));
}

bool newDevPlane(DevNode** out, DevNode** const root, const char* const Name, const D3DMATERIAL9 Material)
{
	if (!mallocDev(out))
		return false;

	newPlane<<<1, 1>>>(out, root, Name, Material);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}