#include "hip/hip_runtime.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "Plane.cuh"

__device__
DevPlane::DevPlane(DevNode** const root, const sp Color)
	: DevNode(root, ePLANE, Color)
{

}

__device__
bool DevPlane::IsInside(const sp& L) const
{
	return (L.y >= 0.0);
}

__device__
bool DevPlane::GetInfo(const sp& K, const sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide) const
{
	if (pHint && pHint->pNode == this && fromOutSide)
		return false;

	float	t = (K.y) ? -L.y / K.y : ((L.y > 0) ? FLT_MAX : -FLT_MAX);

	if (t <= 0)
		return false;

	info.Cross = K * t + L;
	info.Vertical = sp(0, 1, 0);
	info.Distance = t * sqrt(K * K);
	info.isEnter = (L.y < 0);
	info.Material = GetPixel(info.Cross.x, info.Cross.z).getMaterial();
	info.pNode = this;

	return -1 < info.Cross.x && info.Cross.x < 1 && -1 < info.Cross.z && info.Cross.z < 1;
}

__global__
void newPlane(DevNode** out, DevNode** const root, const D3DMATERIAL9 Material)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		*out = new DevPlane(root, sp(Material));
}

bool newDevPlane(DevNode*** out, DevNode** const root, const D3DMATERIAL9 Material)
{
	if (!mallocDev(out))
		return false;

	newPlane<<<1, 1>>>(*out, root, Material);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}