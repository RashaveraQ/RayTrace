#include "hip/hip_runtime.h"
﻿#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "Minus.cuh"

__device__
bool DevMinus::IsInside(const Sp& L) const
{
	return ((m_Member >= 1 && m_Node[0]->IsInside2(L)) && !(m_Member >= 2 && m_Node[1]->IsInside2(L)));
}

__device__
bool DevMinus::GetInfo(const Sp& K, const Sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide) const
{
	DevInfo	l_info;
	DevInfo	r_info;
	Sp		l = L;
	int		left, right;

	if (!(m_Member >= 1 && m_Node[0]->GetInfo2(K, l, l_info, pHint, fromOutSide)))
		return false;
	do {
		left = (m_Member >= 1) ? m_Node[0]->GetInfo2(K, l, l_info, pHint, fromOutSide) : 0;
		right = (m_Member >= 2) ? m_Node[1]->GetInfo2(K, l, r_info, pHint, fromOutSide) : 0;

		if (left == 0 && right == 0)
			return false;

		// 左が、右より近い場合。
		if (cmp_distance(l_info.Distance, r_info.Distance) < 0) {
			l = l_info.Cross;
			info.Material = l_info.Material;
		}
		else {
			l = r_info.Cross;
			info.Material = r_info.Material;
		}

	} while (!IsInside(m_Matrix * l));

	info.isEnter = 1;
	info.Distance = sqrt((L - l)*(L - l));
	info.Cross = l;

	if (info.Material.Diffuse.r < 0)
		info.Material = m_Material;

	return true;
}

__device__
int	DevMinus::cmp_distance(float a, float b) const
{
	if (a >= 0) {
		if (b >= 0)
			return (a < b) ? -1 : 1;
		else
			return -1;
	}
	else {
		if (b >= 0)
			return 1;
		else
			return 0;
	}
}

__global__
void newMinus(DevNode** out, DevNode** const root, const char* const Name)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		*out = new DevMinus(*root, Name);
}

bool newDevMinus(DevNode** out, DevNode** const root, const char* const Name)
{
	if (!mallocDev(out))
		return false;

	newMinus<<<1, 1>>>(out, root, Name);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}