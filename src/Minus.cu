#include "hip/hip_runtime.h"
﻿#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "Minus.cuh"

__device__
bool DevMinus::IsInside(const sp& L) const
{
	return ((m_Member >= 1 && m_Node[0]->IsInside2(L)) && !(m_Member >= 2 && m_Node[1]->IsInside2(L)));
}

__device__
bool DevMinus::GetInfo(const sp& K, const sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide) const
{
	DevInfo	l_info;
	DevInfo	r_info;
	sp		l = L;
	bool	left, right;

	if (!(m_Member >= 1 && m_Node[0]->GetInfo2(K, l, l_info, pHint, fromOutSide)))
		return false;
	do {
		left = (m_Member >= 1) ? m_Node[0]->GetInfo2(K, l, l_info, pHint, fromOutSide) : 0;
		right = (m_Member >= 2) ? m_Node[1]->GetInfo2(K, l, r_info, pHint, fromOutSide) : 0;

		if (!left && !right)
			return false;

		int cmp = cmp_distance(l_info.Distance, r_info.Distance);

		// 左が、右より近い場合。
		if (left && l_info.Distance > 0 && (!right || cmp < 0)) {
			l = l_info.Cross;
			info.pNode = l_info.pNode;
			info.Vertical = l_info.Vertical;
			info.Material = l_info.Material;
		} else if (right && r_info.Distance > 0 && (!left || cmp > 0)) {
			l = r_info.Cross;
			info.pNode = r_info.pNode;
			info.Vertical = r_info.Vertical;
			info.Material = r_info.Material;
		} else
			return false;

	} while (!IsInside(m_Matrix * l));

	info.Distance = sqrt((L - l)*(L - l));
	info.Cross = l;

	if (info.Material.Diffuse.r < 0)
		info.Material = m_Material;

	return true;
}

__device__
int	DevMinus::cmp_distance(float a, float b) const
{
	if (a >= 0) {
		if (b >= 0)
			return (a < b) ? -1 : 1;
		else
			return -1;
	}
	else {
		if (b >= 0)
			return 1;
		else
			return 0;
	}
}

__global__
void newMinus(DevNode** out, DevNode** const root)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		*out = new DevMinus(root);
}

bool newDevMinus(DevNode*** out, DevNode** const root)
{
	if (!mallocDev(out))
		return false;

	newMinus<<<1, 1>>>(*out, root);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}