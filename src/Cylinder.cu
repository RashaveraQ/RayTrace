#include "hip/hip_runtime.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "Cylinder.cuh"

__device__
DevCylinder::DevCylinder(DevNode** const root, const sp Color)
	: DevNode(root, eCYLINDER, Color)
{

}

__device__
bool DevCylinder::IsInside(const sp& L) const
{
	return (-1 <= L.y && L.y <= 1 && sqrt(L.x*L.x + L.z*L.z) <= 1.0);
}

__device__
bool DevCylinder::GetInfo(const sp& K, const sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide) const
{
	if (pHint && pHint->pNode == this && fromOutSide)
		return false;

	if (L.y < -1) {
		if (K.y <= 0)
			return false;

		float t = -(1 + L.y) / K.y;
		if (t > 0) {
			sp	p = K * t + L;
			if (p.x * p.x + p.z * p.z <= 1) {
				info.Cross = p;
				info.Vertical = sp(0, -1, 0);
				info.Distance = t * sqrt(K * K);
				info.Material = GetPixel(.5f*(p.x + 1), .5f*(p.z + 1)).getMaterial();
				info.pNode = this;
				return true;
			}
		}
	}

	if (L.y > 1) {
		if (K.y >= 0)
			return false;

		float t = (1 - L.y) / K.y;
		if (t > 0) {
			sp	p = K * t + L;
			if (p.x * p.x + p.z * p.z <= 1) {
				info.Cross = p;
				info.Vertical = sp(0, 1, 0);
				info.Distance = t * sqrt(K * K);
				info.Material = GetPixel(.5f*(p.x + 1), .5f*(p.z + 1)).getMaterial();
				info.pNode = this;

				return true;
			}
		}
	}

	float	a, b, c, d, t, t1, t2;

	c = K.x * L.z - K.z * L.x;
	c *= c;
	a = K.x * K.x + K.z * K.z;
	d = a - c;

	if (d < 0)
		return FALSE;

	d = sqrt(d);
	b = -(K.x * L.x + K.z * L.z);

	t1 = (b + d) / a;
	t2 = (b - d) / a;

	if (t1 > 0) {
		if (t2 > 0) {
			t = (t1 < t2) ? t1 : t2;
		}
		else
			t = t1;
	}
	else {
		if (t2 > 0)
			t = t2;
		else
			return false;
	}

	sp p = K * t + L;

	if (p.y < -1 || 1 < p.y)
		return false;

	info.Cross = info.Vertical = p;
	info.Vertical.y = 0;
	info.Distance = t * sqrt(K * K);
	info.Material = m_Material;
	info.pNode = this;

	return true;
}

__global__
void newCylinder(DevNode** out, DevNode** const root, const D3DMATERIAL9 Material)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		*out = new DevCylinder(root, sp(Material));
}

bool newDevCylinder(DevNode*** out, DevNode** const root, const D3DMATERIAL9 Material)
{
	if (!mallocDev(out))
		return false;

	newCylinder<<<1, 1>>>(*out, root, Material);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}