#include "hip/hip_runtime.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "DoCuda.h"
#include "Cube.cuh"

__device__
DevCube::DevCube(DevNode** const root, const sp Color)
	: DevNode(root, CUBE, Color)
{

}

__device__
bool DevCube::IsInside(const sp& L) const
{
	return (-1.0 <= L.x && L.x <= 1.0 && -1.0 <= L.y && L.y <= 1.0 && -1.0 <= L.z && L.z <= 1.0);
}

__device__
bool DevCube::GetInfo(const sp& K, const sp& L, DevInfo& info, const DevInfo* pHint, bool fromOutSide) const
{
	if (pHint && pHint->pNode == this && fromOutSide)
		return false;

	float t[6];
	sp	   v[6];
	int i = 0;

	info.isEnter = (pHint && pHint->pNode == this) ? fromOutSide : !IsInside(L);

	t[i] = (-1 - L.x) / K.x;
	if (t[i] > 0) {
		sp p = K * t[i] + L;
		if (-1 <= p.y && p.y <= 1 && -1 <= p.z && p.z <= 1) {
			v[i] = sp(-1, 0, 0);
			i++;
		}
	}

	t[i] = (1 - L.x) / K.x;
	if (t[i] > 0) {
		sp p = K * t[i] + L;
		if (-1 <= p.y && p.y <= 1 && -1 <= p.z && p.z <= 1) {
			v[i] = sp(1, 0, 0);
			i++;
		}
	}

	t[i] = (-1 - L.y) / K.y;
	if (t[i] > 0) {
		sp p = K * t[i] + L;
		if (-1 <= p.x && p.x <= 1 && -1 <= p.z && p.z <= 1) {
			v[i] = sp(0, -1, 0);
			i++;
		}
	}

	t[i] = (1 - L.y) / K.y;
	if (t[i] > 0) {
		sp p = K * t[i] + L;
		if (-1 <= p.x && p.x <= 1 && -1 <= p.z && p.z <= 1) {
			v[i] = sp(0, 1, 0);
			i++;
		}
	}

	t[i] = (-1 - L.z) / K.z;
	if (t[i] > 0) {
		sp p = K * t[i] + L;
		if (-1 <= p.y && p.y <= 1 && -1 <= p.x && p.x <= 1) {
			v[i] = sp(0, 0, -1);
			i++;
		}
	}

	t[i] = (1 - L.z) / K.z;
	if (t[i] > 0) {
		sp p = K * t[i] + L;
		if (-1 <= p.y && p.y <= 1 && -1 <= p.x && p.x <= 1) {
			v[i] = sp(0, 0, 1);
			i++;
		}
	}

	switch (i) {
	case 0:
		return false;
	case 1:
		break;
	default:
		if (pHint && pHint->pNode == this) {
			if (t[0] < t[1]) {
				t[0] = t[1];
				v[0] = v[1];
			}
		}
		else if (t[0] > t[1]) {
			t[0] = t[1];
			v[0] = v[1];
		}
		break;
	}

	info.Cross = K * t[0] + L;
	info.Vertical = info.isEnter ? v[0] : -v[0];
	info.Distance = t[0] * sqrt(K * K);
	info.Material = m_Material;
	info.pNode = this;
	return true;
}

__global__
void newCube(DevNode** out, DevNode** const root, const D3DMATERIAL9 Material)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		*out = new DevCube(root, sp(Material));
}

bool newDevCube(DevNode*** out, DevNode** const root, const D3DMATERIAL9 Material)
{
	if (!mallocDev(out))
		return false;

	newCube<<<1, 1>>>(*out, root, Material);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		return false;
	}

	return true;
}