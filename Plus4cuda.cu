__device__
void GetInfo_Plus(const Task& task, const sp4cuda& K, const sp4cuda& L, Info4cuda& info, Stack& stack)
{
	double	l = -1;
	for (int i = 0; i < task.data.gathering.member; i++) {
		int j = --stack.Index;
		if (stack.valid[j]) {
			if (l == -1 || stack.Distance[j] < l) {
				l = stack.Distance[j];
				info.valid		= stack.valid	[j];
				info.Material	= stack.Material[j];
				info.isEnter	= stack.isEnter	[j];	// ���荞��
				info.Distance	= stack.Distance[j];	// ��_�܂ł̋���
				info.Cross_x	= stack.Cross_x[j];		// ��_���Wx
				info.Cross_y	= stack.Cross_y[j];		// ��_���Wy
				info.Cross_z	= stack.Cross_z[j];		// ��_���Wz
				info.Vertical_x	= stack.Vertical_x[j];	// �@���x�N�g��x
				info.Vertical_y	= stack.Vertical_y[j];	// �@���x�N�g��y
				info.Vertical_z	= stack.Vertical_z[j];	// �@���x�N�g��z
				info.Refractive	= stack.Refractive[j];	// ���ܗ�
				info.nodeInfo	= stack.nodeInfo[j];	//
			}
		}
	}

	if (l < 0) {
		info.valid = false;
		return;
	}

	if (info.Material.Diffuse.r < 0)
		info.Material = task.nodeInfo.m_Material;
	info.valid = true;
}
