extern "C" __device__ 
void GetInfo_Plus(Task task, sp4cuda K, sp4cuda L, Info4cuda* info, Stack *stack)
{
	double	l = -1;
	for (int i = 0; i < task.data.gathering.member; i++) {
		int j = --(stack->Index);
		if (stack->valid[j]) {
			if (l == -1 || stack->Distance[j] < l) {
				l = stack->Distance[j];
				info->valid			= stack->valid[j];
				info->Color_r		= stack->Color_r[j];
				info->Color_b		= stack->Color_g[j];
				info->Color_g		= stack->Color_b[j];
				info->isEnter		= stack->isEnter[j];	// ���荞��
				info->Distance		= stack->Distance[j];	// ��_�܂ł̋���
				info->Cross_x		= stack->Cross_x[j];		// ��_���Wx
				info->Cross_y		= stack->Cross_y[j];		// ��_���Wy
				info->Cross_z		= stack->Cross_z[j];		// ��_���Wz
				info->Vertical_x	= stack->Vertical_x[j];	// �@���x�N�g��x
				info->Vertical_y	= stack->Vertical_y[j];	// �@���x�N�g��y
				info->Vertical_z	= stack->Vertical_z[j];	// �@���x�N�g��z
				info->Refractive	= stack->Refractive[j];	// ���ܗ�
				info->nodeInfo		= stack->nodeInfo[j];	//
			}
		}
	}

	if (l < 0) {
		info->valid = false;
		return;
	}

//	if (info.Material.Diffuse.r < 0)
	if (info->Color_r < 0) {
		info->Color_r = task.nodeInfo.m_Color_r;
		info->Color_g = task.nodeInfo.m_Color_g;
		info->Color_b = task.nodeInfo.m_Color_b;
	}
	info->valid = true;
}
