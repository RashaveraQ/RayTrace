extern "C" __device__ 
Info4cuda GetInfo_Plus(Task task, sp4cuda K, sp4cuda L, Stack *stack)
{
	int idx = stack->Index;
	Info4cuda info;

return info;
	float	l = -1;
	for (int i = 0; i < task.data.gathering.member; i++) {
		int j = --idx;
		if (stack->data[j].valid) {
			if (l == -1 || stack->data[j].Distance < l) {
				StackData* sd = &(stack->data[j]);
				l = sd->Distance;
				info.valid		= sd->valid;
				info.Color_r	= sd->Color_r;
				info.Color_b	= sd->Color_g;
				info.Color_g	= sd->Color_b;
				info.isEnter	= sd->isEnter;		// ���荞��
				info.Distance	= sd->Distance;		// ��_�܂ł̋���
				info.Cross_x	= sd->Cross_x;		// ��_���Wx
				info.Cross_y	= sd->Cross_y;		// ��_���Wy
				info.Cross_z	= sd->Cross_z;		// ��_���Wz
				info.Vertical_x	= sd->Vertical_x;	// �@���x�N�g��x
				info.Vertical_y	= sd->Vertical_y;	// �@���x�N�g��y
				info.Vertical_z	= sd->Vertical_z;	// �@���x�N�g��z
				info.Refractive	= sd->Refractive;	// ���ܗ�
				info.nodeInfo	= sd->nodeInfo;		//
			}
		}
	}

	if (l < 0) {
		info.valid = false;
		return info;
	}

//	if (info.Material.Diffuse.r < 0)
	if (info.Color_r < 0) {
		info.Color_r = task.nodeInfo.m_Color_r;
		info.Color_g = task.nodeInfo.m_Color_g;
		info.Color_b = task.nodeInfo.m_Color_b;
	}
	info.valid = true;
	
	return info;
}
