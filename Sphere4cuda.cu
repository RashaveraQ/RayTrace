#include "hip/hip_runtime.h"

extern "C" __device__
Info4cuda GetInfo_Sphere(Task task, sp4cuda K, sp4cuda L)
{
	Info4cuda info;

	float	a = sp_internal_multiple(K, K);
	float	b = sp_internal_multiple(K, L);
	float	c = sp_internal_multiple(L, L) - 1.0; 

	float	bb_ac = b * b - a * c;

	if (bb_ac < 0) {
		info.valid = false;
		return info;
	}

	float	t;
	float	t1 = (-b + sqrtf(bb_ac)) / a;
	float	t2 = (-b - sqrtf(bb_ac)) / a;

	info.isEnter = 0;
	if (t1 > 0) {
		if (t2 > 0) {
			t = (t1 < t2) ? t1 : t2;
			info.isEnter = 1;
		} else
			t = t1;
	} else {
		if (t2 > 0)
			t = t2;
		else {
			info.valid = false;
			return info;
		}
	}

	sp4cuda cross = sp_plus(sp_multiple(K, t), L);
	info.Cross_x = info.Vertical_x = cross.x;
	info.Cross_y = info.Vertical_y = cross.y;
	info.Cross_z = info.Vertical_z = cross.z;
	
	info.Distance = t * sqrtf(sp_internal_multiple(K,K));

/*
	float x, y, z, th, phy;
	x = info.Vertical_x;
	y = info.Vertical_y;
	z = info.Vertical_z;
	float th, phy
	th = atan2(y, sqrt(x*x+z*z)) / M_PI + .5;
	phy = atan2(x, -z) / (2 * M_PI) + .5;
	info.Material = GetPixel(phy, th).getMaterial();
*/
//	info.Material = sp_getMaterial(sp_sp(256 * task.nodeInfo.m_Material.Diffuse.r, 256 * task.nodeInfo.m_Material.Diffuse.g, 256 * task.nodeInfo.m_Material.Diffuse.b));
	info.Color_r = task.nodeInfo.m_Color_r;
	info.Color_g = task.nodeInfo.m_Color_g;
	info.Color_b = task.nodeInfo.m_Color_b;
	info.nodeInfo = task.nodeInfo;
	info.valid = true;
	
	return info;
}
