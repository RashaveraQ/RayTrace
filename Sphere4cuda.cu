#include "hip/hip_runtime.h"
__device__
void GetInfo_Sphere(const Task& task, const sp& K, const sp& L, Info4cuda& info)
{
	double	a = K * K;
	double	b = K * L;
	double	c = L * L - 1.0; 

	double	bb_ac = b*b - a*c;

	if (bb_ac < 0) {
		info.valid = false;
		return;
	}

	double	t;
	double	t1 = (-b + sqrt(bb_ac)) / a;
	double	t2 = (-b - sqrt(bb_ac)) / a;

	info.isEnter = 0;
	if (t1 > 0) {
		if (t2 > 0) {
			t = (t1 < t2) ? t1 : t2;
			info.isEnter = 1;
		} else
			t = t1;
	} else {
		if (t2 > 0)
			t = t2;
		else {
			info.valid = false;
			return;
		}
	}

	sp cross = K * t + L;
	info.Cross_x = info.Vertical_x = cross.x;
	info.Cross_y = info.Vertical_y = cross.y;
	info.Cross_z = info.Vertical_z = cross.z;
	
	info.Distance = t * sqrt(K * K);

	double x,y,z, th, phy;

	x = info.Vertical_x;
	y = info.Vertical_y;
	z = info.Vertical_z;

	th = atan2(y, sqrt(x*x+z*z)) / M_PI + .5;
	phy = atan2(x, -z) / (2 * M_PI) + .5;

	//info.Material = GetPixel(phy, th).getMaterial();
	info.Material = sp(	256 * task.nodeInfo.m_Material.Diffuse.r,
						256 * task.nodeInfo.m_Material.Diffuse.g,
						256 * task.nodeInfo.m_Material.Diffuse.b).getMaterial();
	info.nodeInfo = task.nodeInfo;
	info.valid = true;
}
