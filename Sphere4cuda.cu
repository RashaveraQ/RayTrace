#include "hip/hip_runtime.h"

extern "C" __device__
void GetInfo_Sphere(Task task, sp4cuda K, sp4cuda L, Info4cuda *info)
{
	double	a = sp_internal_multiple(K, K);
	double	b = sp_internal_multiple(K, L);
	double	c = sp_internal_multiple(L, L) - 1.0; 

	double	bb_ac = b * b - a * c;

	if (bb_ac < 0) {
		info->valid = false;
		return;
	}

	double	t;
	double	t1 = (-b + sqrt(bb_ac)) / a;
	double	t2 = (-b - sqrt(bb_ac)) / a;

	info->isEnter = 0;
	if (t1 > 0) {
		if (t2 > 0) {
			t = (t1 < t2) ? t1 : t2;
			info->isEnter = 1;
		} else
			t = t1;
	} else {
		if (t2 > 0)
			t = t2;
		else {
			info->valid = false;
			return;
		}
	}

	sp4cuda cross = sp_plus(sp_multiple(K, t), L);
	info->Cross_x = info->Vertical_x = cross.x;
	info->Cross_y = info->Vertical_y = cross.y;
	info->Cross_z = info->Vertical_z = cross.z;
	
	info->Distance = t * sqrt(sp_internal_multiple(K,K));

/*
	double x, y, z, th, phy;
	x = info.Vertical_x;
	y = info.Vertical_y;
	z = info.Vertical_z;
	double th, phy
	th = atan2(y, sqrt(x*x+z*z)) / M_PI + .5;
	phy = atan2(x, -z) / (2 * M_PI) + .5;
	info.Material = GetPixel(phy, th).getMaterial();
*/
//	info.Material = sp_getMaterial(sp_sp(256 * task.nodeInfo.m_Material.Diffuse.r, 256 * task.nodeInfo.m_Material.Diffuse.g, 256 * task.nodeInfo.m_Material.Diffuse.b));
	info->Color_r = task.nodeInfo.m_Color_r;
	info->Color_g = task.nodeInfo.m_Color_g;
	info->Color_b = task.nodeInfo.m_Color_b;
	info->nodeInfo = task.nodeInfo;
	info->valid = true;
}
