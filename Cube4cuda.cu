#include "hip/hip_runtime.h"
__device__
void GetInfo_Cube(const Task& task, const sp& K, const sp& L, Info4cuda& info)
{
	info.isEnter = (-1.0 > L.x || L.x > 1.0 || -1.0 > L.y || L.y > 1.0 || -1.0 > L.z || L.z > 1.0);

	if (info.isEnter) {
		//�O������O�Ɍ������ꍇ�A
		if ((L.x < -1.0 && K.x <= 0.0) || (1.0 < L.x && 0.0 <= K.x) ||
			(L.y < -1.0 && K.y <= 0.0) || (1.0 < L.y && 0.0 <= K.y) ||
			(L.z < -1.0 && K.z <= 0.0) || (1.0 < L.z && 0.0 <= K.z)) {
			info.valid = false;
			return;
		}
	}

	int		i, j, k;
	double	T[6], t1, t;
	sp		c;

	// �S�Ă̖ʂ܂ł̋��������߂�B
	T[0] = (1.0 - L.x) / K.x; T[1] = - (1.0 + L.x) / K.x;
	T[2] = (1.0 - L.y) / K.y; T[3] = - (1.0 + L.y) / K.y;
	T[4] = (1.0 - L.z) / K.z; T[5] = - (1.0 + L.z) / K.z;

	// t1 �ɍő�l��������B
	for (i = 1, t1 = T[0]; i < 6; i++)
		if (t1 < T[i])
			t1 = T[i];

	// ���̏ꍇ�At1 ��������B
	for (i = 0; i < 6; i++)
		if (T[i] < 0.0)
			T[i] = t1;

	for (k = 0; k < 3; k++)	{
		for (i = 1, j = 0, t= T[0]; i < 6; i++)
			if (t > T[i])
				t = T[i], j = i;

		if (!info.isEnter)
			break;

		c = K * t + L;

		if (-1.0 <= c.x && c.x <= 1.0 && -1.0 <= c.y && c.y <= 1.0 && -1.0 <= c.z && c.z <= 1.0)
			break;

		T[j] = t1;
	}

	if (k == 3) {
		info.valid = false;
		return;
	}

	sp cross = K * t + L;
	info.Cross_x = cross.x;
	info.Cross_y = cross.y;
	info.Cross_z = cross.z;
	info.Distance = t * sqrt(K * K);
	info.Material = task.nodeInfo.m_Material;

	switch (j) {
	case 0:
		info.Vertical_x = 1;
		info.Vertical_y = 0;
		info.Vertical_z = 0;
		break;
	case 1:
		info.Vertical_x = -1;
		info.Vertical_y = 0;
		info.Vertical_z = 0;
		break;
	case 2:
		info.Vertical_x = 0;
		info.Vertical_y = 1;
		info.Vertical_z = 0;
		break;
	case 3:
		info.Vertical_x = 0;
		info.Vertical_y = -1;
		info.Vertical_z = 0;
		break;
	case 4:
		info.Vertical_x = 0;
		info.Vertical_y = 0;
		info.Vertical_z = 1;
		break;
	case 5:
		info.Vertical_x = 0;
		info.Vertical_y = 0;
		info.Vertical_z = -1;
		break;
	}
	info.nodeInfo = task.nodeInfo;
	info.valid = true;
}
