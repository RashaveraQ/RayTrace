#include "hip/hip_runtime.h"
__device__
void GetInfo_Cylinder(const Task& task, const sp& K, const sp& L, Info4cuda& info)
{
	if (L.y < -1) {
		if (K.y <= 0) {
			info.valid = false;
			return;
		}

		float t = -(1 + L.y) / K.y;
		sp	p = K * t + L;

		if (p.x * p.x + p.z * p.z <= 1) {
			info.Cross_x = p.x;
			info.Cross_y = p.y;
			info.Cross_z = p.z;
			info.Vertical_x = 0;
			info.Vertical_y = -1;
			info.Vertical_z = 0;
			info.Distance = t * sqrt(K * K);
			info.isEnter = 1;
//			info.Material = GetPixel(.5*(p.x+1),.5*(p.z+1)).getMaterial();
			info.Material = sp(256 * task.nodeInfo.m_Material.Diffuse.r,
							   256 * task.nodeInfo.m_Material.Diffuse.g,
							   256 * task.nodeInfo.m_Material.Diffuse.b).getMaterial();
			info.nodeInfo = task.nodeInfo;
			info.valid = true;
			return;
		}
	}

	if (L.y > 1) {
		if (K.y >= 0) {
			info.valid = false;
			return;
		}

		float t = (1 - L.y) / K.y;

		sp	p = K*t+L;

		if (p.x * p.x + p.z * p.z <= 1) {
			info.Cross_x = p.x;
			info.Cross_y = p.y;
			info.Cross_z = p.z;
			info.Vertical_x = 0;
			info.Vertical_y = 1;
			info.Vertical_z = 0;
			info.Distance = t * sqrt(K * K);
			info.isEnter = 1;
//			info.Material = GetPixel(.5*(p.x+1),.5*(p.z+1)).getMaterial();
			info.Material = sp(256 * task.nodeInfo.m_Material.Diffuse.r,
							   256 * task.nodeInfo.m_Material.Diffuse.g,
							   256 * task.nodeInfo.m_Material.Diffuse.b).getMaterial();
			info.nodeInfo = task.nodeInfo;
			info.valid = true;
			return;
		}
	}

	float	a, b, c, d, t, t1, t2;

	c = K.x * L.z - K.z * L.x;
	c *= c;
	a = K.x * K.x + K.z * K.z;
	d = a - c;

	if (d < 0) {
		info.valid = false;
		return;
	}

	d = sqrt(d);
	b = -(K.x * L.x + K.z * L.z);

	t1 = (b + d) / a;
	t2 = (b - d) / a;

	info.isEnter = 0;
	if (t1 > 0) {
		if (t2 > 0) {
			t = (t1 < t2) ? t1 : t2;
			info.isEnter = 1;
		} else
			t = t1;
	} else {
		if (t2 > 0)
			t = t2;
		else {
			info.valid = false;
			return;
		}
	}

	sp p = K * t + L;

	if (p.y < -1 || 1 < p.y) {
		info.valid = false;
		return;
	}

	info.isEnter = (-1 > L.y || L.y > 1 || sqrt(L.x*L.x+L.z*L.z) > 1.0);

	info.Cross_x = info.Vertical_x = p.x;
	info.Cross_y = info.Vertical_y = p.y;
	info.Cross_z = info.Vertical_z = p.z;
	info.Vertical_y = 0;
	info.Distance = t * sqrt(K * K);
	info.Material = task.nodeInfo.m_Material;
	info.nodeInfo = task.nodeInfo;
	info.valid = true;
}
