#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cutil4win.h"
#include <d3dx9.h>
#include <atltypes.h>

#define TARGET __device__
#include "task.h"
#include "sp.h"
#include "matrix.h"

#ifndef M_PI
#define M_PI (4.0*atan(1.0))
#endif

__constant__ Task cTask[100];
__constant__ int  cTaskIndex;
static int sTaskIndex = 0;

struct Info4cuda
{
	bool			valid;
	D3DMATERIAL9	Material;
	BOOL			isEnter;	// ���荞��
	double			Distance;	// ��_�܂ł̋���
	double			Cross_x;	// ��_���Wx
	double			Cross_y;	// ��_���Wy
	double			Cross_z;	// ��_���Wz
	double			Vertical_x;	// �@���x�N�g��x
	double			Vertical_y;	// �@���x�N�g��y
	double			Vertical_z;	// �@���x�N�g��z
	double			Refractive;	// ���ܗ�
	NodeInfo		nodeInfo;	//
};

#define STACK_SIZE	50

struct Stack {
	int				Index;
	bool			valid		[STACK_SIZE];
	D3DMATERIAL9	Material	[STACK_SIZE];
	BOOL			isEnter		[STACK_SIZE];	// ���荞��
	double			Distance	[STACK_SIZE];	// ��_�܂ł̋���
	double			Cross_x		[STACK_SIZE];	// ��_���Wx
	double			Cross_y		[STACK_SIZE];	// ��_���Wy
	double			Cross_z		[STACK_SIZE];	// ��_���Wz
	double			Vertical_x	[STACK_SIZE];	// �@���x�N�g��x
	double			Vertical_y	[STACK_SIZE];	// �@���x�N�g��y
	double			Vertical_z	[STACK_SIZE];	// �@���x�N�g��z
	double			Refractive	[STACK_SIZE];	// ���ܗ�
	NodeInfo		nodeInfo	[STACK_SIZE];	//
};

__device__
void GetInfo_Sphere(const Task& task, const sp& K, const sp& L, Info4cuda& info)
{
	double	a = K * K;
	double	b = K * L;
	double	c = L * L - 1.0; 

	double	bb_ac = b*b - a*c;

	if (bb_ac < 0) {
		info.valid = false;
		return;
	}

	double	t;
	double	t1 = (-b + sqrt(bb_ac)) / a;
	double	t2 = (-b - sqrt(bb_ac)) / a;

	info.isEnter = 0;
	if (t1 > 0) {
		if (t2 > 0) {
			t = (t1 < t2) ? t1 : t2;
			info.isEnter = 1;
		} else
			t = t1;
	} else {
		if (t2 > 0)
			t = t2;
		else {
			info.valid = false;
			return;
		}
	}

	sp cross = K * t + L;
	info.Cross_x = info.Vertical_x = cross.x;
	info.Cross_y = info.Vertical_y = cross.y;
	info.Cross_z = info.Vertical_z = cross.z;
	
	info.Distance = t * sqrt(K * K);

	double x,y,z, th, phy;

	x = info.Vertical_x;
	y = info.Vertical_y;
	z = info.Vertical_z;

	th = atan2(y, sqrt(x*x+z*z)) / M_PI + .5;
	phy = atan2(x, -z) / (2 * M_PI) + .5;

	//info.Material = GetPixel(phy, th).getMaterial();
	info.Material = sp(	256 * task.nodeInfo.m_Material.Diffuse.r,
						256 * task.nodeInfo.m_Material.Diffuse.g,
						256 * task.nodeInfo.m_Material.Diffuse.b).getMaterial();
	info.nodeInfo = task.nodeInfo;
	info.valid = true;
}

__device__
void GetInfo_Plus(const Task& task, const sp& K, const sp& L, Info4cuda& info, Stack& stack)
{
	Info4cuda	tmp;
	double	l = -1;

	for (int i = 0; i < task.member; i++) {
		int j = stack.Index--;
		//if (m_Node[i]->GetInfo2(K, L, tmp)) {
		if (stack.valid[j]) {
			if (l == -1 || stack.Distance[j] < l) {
				l = stack.Distance[j];
				info.valid		= stack.valid	[j];
				info.Material	= stack.Material[j];
				info.isEnter	= stack.isEnter	[j];	// ���荞��
				info.Distance	= stack.Distance[j];	// ��_�܂ł̋���
				info.Cross_x	= stack.Cross_x[j];		// ��_���Wx
				info.Cross_y	= stack.Cross_y[j];		// ��_���Wy
				info.Cross_z	= stack.Cross_z[j];		// ��_���Wz
				info.Vertical_x	= stack.Vertical_x[j];	// �@���x�N�g��x
				info.Vertical_y	= stack.Vertical_y[j];	// �@���x�N�g��y
				info.Vertical_z	= stack.Vertical_z[j];	// �@���x�N�g��z
				info.Refractive	= stack.Refractive[j];	// ���ܗ�
				info.nodeInfo	= stack.nodeInfo[j];	//
			}
		}
	}

	if (l < 0) {
		info.valid = false;
		return;
	}

	if (info.Material.Diffuse.r < 0)
		info.Material = task.nodeInfo.m_Material;

	info.valid = true;
}

__device__
bool GetInfo2(const sp& K, const sp& L, Info4cuda& info)
{
	Stack stack;

	for (int idx = 0; idx < cTaskIndex; idx++) {
		Info4cuda	inf;
		
		matrix m;
		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				m.m_data[i][j] = cTask[idx].m[i][j];
			}
		}
		matrix Inv_m = m.Inv();

		sp L2 = Inv_m * L;
		sp K2 = Inv_m * (K + L) - L2;

		switch (cTask[idx].type) {
		case SPHERE:
			GetInfo_Sphere(cTask[idx], K2, L2, inf);
			break;
		case PLANE:
			break;
		case PLUS:
			GetInfo_Plus(cTask[idx], K2, L2, inf, stack);
			break;
		case MINUS:
			break;
		case MULTIPLE:
			break;
		case CONE:
			break;
		case CYLINDER:
			break;
		case TORUS:
			break;
		case POLYGON:
			break;
		case CUBE:
			break;
		case TEAPOT:
			break;
		}
		
		// inf ���X�^�b�N�ɐςށB
		stack.valid			[stack.Index] = inf.valid;
		stack.Material		[stack.Index] = inf.Material;
		stack.isEnter		[stack.Index] = inf.isEnter;	// ���荞��
		stack.Distance		[stack.Index] = inf.Distance;	// ��_�܂ł̋���
		stack.Cross_x		[stack.Index] = inf.Cross_x;	// ��_���Wx
		stack.Cross_y		[stack.Index] = inf.Cross_y;	// ��_���Wy
		stack.Cross_z		[stack.Index] = inf.Cross_z;	// ��_���Wz
		stack.Vertical_x	[stack.Index] = inf.Vertical_x;	// �@���x�N�g��x
		stack.Vertical_y	[stack.Index] = inf.Vertical_y;	// �@���x�N�g��y
		stack.Vertical_z	[stack.Index] = inf.Vertical_z;	// �@���x�N�g��z
		stack.Refractive	[stack.Index] = inf.Refractive;	// ���ܗ�
		stack.nodeInfo		[stack.Index] = inf.nodeInfo;	//
		stack.Index++;
	}

	bool ans = stack.valid[0];

	return ans;
}

__device__
sp GetColor(sp K, sp L, const sp& Light)
{
	sp ans(127,127,127);

	enum eMode {
		eNone, eReflect, eThrough
	};
	eMode mode = eNone;

	Info4cuda info;
	
	for (int i = 0; i < 10; i++) {
		
		switch (mode) {
		case eNone:
			break;
		case eReflect:
			// ���˗��ŐF��������B
			info.Material = (info.nodeInfo.m_Reflect * ans + (1 - info.nodeInfo.m_Reflect) * sp(info.Material)).getMaterial();
			break;
		case eThrough:
			// ���ߗ��ŐF��������B
			info.Material = (info.nodeInfo.m_Through * ans + (1 - info.nodeInfo.m_Through) * sp(info.Material)).getMaterial();
			break;
		}

		if (!GetInfo2(K, L, info))
			break;
			
		sp k = K.e();
		sp v(info.Vertical_x, info.Vertical_y, info.Vertical_y);

		v = v.e();

		// ���˗�������ꍇ�A
		if (info.nodeInfo.m_Reflect > 0) {
			sp k2 = k - 2 * (v * k) * v;
			sp cross(info.Cross_x, info.Cross_y, info.Cross_z);
			sp l2 = cross + 1E-05 * k2;
			// ���˂��������x�N�g������F���擾�B
			K = k2, L = l2, mode = eReflect;
			continue;
		}

		// ���ߗ�������ꍇ�A
		if (info.nodeInfo.m_Through > 0) {
			double r = info.Refractive;
			double i = k * v;
			sp k2 = r * (k -i * v - sqrt(r * r - 1.0 + i * i) * v);
			sp cross(info.Cross_x, info.Cross_y, info.Cross_z);
			sp l2 = cross + 1E-05 * k2;
			// ���܂��������x�N�g������F���擾�B
			K = k2, L = l2, mode = eThrough;
			continue;
		}

		// �������F��␳�B
		sp vertical(info.Vertical_x, info.Vertical_y, info.Vertical_z);
		
		double	x = -Light.e() * vertical.e();
		x = (x > 0.0) ? x : 0.0;
		double t = 64 + 191 * sin(M_PI / 2 * x);
		double b = 191 * (1 - cos(M_PI / 2 * x));
		
		ans =  (t - b) * sp(info.Material) / 255 + sp(b,b,b);
	}

	return ans;
}

__global__
void kernel(unsigned long* dst, int imageW, int imageH, matrix* pMatrix, sp* pLight)
{
    const int px = blockDim.x * blockIdx.x + threadIdx.x;
    const int py = blockDim.y * blockIdx.y + threadIdx.y;
   	double rx = 20.0 * px / imageW - 10.0;
	double ry = 20.0 * py / imageH - 10.0;

	sp k(0.01 * rx / 20.0, 0.01 * ry / 20.0, 0.01);
	sp l(rx, ry, -20);
    
	k = *pMatrix * (k + l) - *pMatrix * l;
	l = *pMatrix * l;

	sp c = GetColor(k, l, *pLight);
//	sp c((imageW * cTaskIndex) % 256, (imageH * cTaskIndex) % 256, 0);

	if (px <= imageW && py <= imageH) {
		dst[px + py * imageW] = RGB(c.x, c.y, c.z);
	}
}

void DoCuda(unsigned long* out, const int imageW, const int imageH, const matrix* m, const sp* light)
{
	matrix*		pMatrix;
	sp*			pLight;
	unsigned long* d_data;
    unsigned int mem_size = imageW * imageH * sizeof(unsigned long);
    

    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, mem_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&pMatrix, sizeof(matrix)));
	CUDA_SAFE_CALL(hipMalloc((void**)&pLight, sizeof(sp)));

	CUDA_SAFE_CALL(hipMemcpy(pMatrix, m, sizeof(matrix), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(pLight, light, sizeof(sp), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol<int>(cTaskIndex, &sTaskIndex, sizeof(int)));

    dim3 threads(16,16);
    dim3 grid(16,16);
//    dim3 grid(iDivUp(imageW, 16), iDivUp(imageH, 16));

	// execute the kernel
	kernel<<< grid, threads >>>(d_data, imageW, imageH, pMatrix, pLight);
	CUT_CHECK_ERROR("kernel failed.");

    // copy results from device to host
    CUDA_SAFE_CALL(hipMemcpy(out, d_data, mem_size, hipMemcpyDeviceToHost));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_data));
    CUDA_SAFE_CALL(hipFree(pMatrix));
    CUDA_SAFE_CALL(hipFree(pLight));
}

void ClearTask()
{
	sTaskIndex = 0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol<int>(cTaskIndex, &sTaskIndex, sizeof(int)));
}

void AddTask(const Task& task)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol<Task>(cTask[0], &task, sizeof(Task), sTaskIndex * sizeof(Task)));
	sTaskIndex++;
}
