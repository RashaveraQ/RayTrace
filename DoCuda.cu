#include "hip/hip_runtime.h"
#include <cutil.h>
#include <hip/hip_runtime.h>
#include <d3dx9.h>
#include <atltypes.h>

#define TARGET __device__
#include "matrix.h"

// The dimensions of the thread block
#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16

__device__ sp GetColor(void* target, const sp* pK, const sp* pL, int nest)
{
	sp c(255, 255 * sin(pL->x), 255 * sin(pL->y));
	return c;
}

__global__ void kernel(unsigned long* dst, const int imageW, const int imageH, void* root, const matrix* m)
{
    const int px = blockDim.x * blockIdx.x + threadIdx.x;
    const int py = blockDim.y * blockIdx.y + threadIdx.y;
   	double rx = 20.0 * px / imageW - 10.0;
	double ry = 20.0 * py / imageH - 10.0;

	sp k(0.01 * rx / 20.0, 0.01 * ry / 20.0, 0.01);
	sp l(rx, ry, -20);
    
	k = *m * (k + l) - *m * l;
	l = *m * l;

	sp c = GetColor(root, &k, &l, 0);
	
	if (px <= imageW && py <= imageH) {
		dst[px + py * imageW] = RGB(c.x, c.y, c.z);
	}
}

void DoCuda(unsigned long* out, class Node* root, const int imageW, const int imageH, const matrix* m)
{
	hipError_t err;

	unsigned long* d_data;
    unsigned int mem_size = imageW * imageH * sizeof(unsigned long);
	if (hipSuccess != (err = hipMalloc((void**)&d_data, mem_size))) {
		MessageBox(0, hipGetErrorString(err), "hipMalloc:1", MB_OK);
		return;
	}

	matrix* d_m;
	if (hipSuccess != (err = hipMalloc((void**)&d_m, sizeof(matrix)))) {
		MessageBox(0, hipGetErrorString(err), "hipMalloc:2", MB_OK);
		return;
	}	
    if (hipSuccess != (err = hipMemcpy(d_m, m, sizeof(matrix), hipMemcpyHostToDevice))) {
		MessageBox(0, hipGetErrorString(err), "hipMemcpy:1", MB_OK);
	}
	
    dim3 threads(16,16);
    dim3 grid(16,16);
 
	// execute the kernel
	kernel<<< grid, threads >>>(d_data, imageW, imageH, root, d_m);

    if (hipSuccess != hipGetLastError()) {
		MessageBox(0, hipGetErrorString(err), "hipGetLastError", MB_OK);
		return;
	}
	
    if (hipSuccess != (err = hipDeviceSynchronize())) {
		MessageBox(0, hipGetErrorString(err), "hipDeviceSynchronize", MB_OK);
    	return;
    }

    // copy results from device to host
    if (hipSuccess != (err = hipMemcpy(out, d_data, mem_size, hipMemcpyDeviceToHost))) {
		MessageBox(0, hipGetErrorString(err), "hipMemcpy:2", MB_OK);
	}

    // cleanup memory
    if (hipSuccess != (err = hipFree(d_data))) {
		MessageBox(0, hipGetErrorString(err), "hipFree", MB_OK);
		return;
	}
}

