#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cutil4win.h"
//#include <atltypes.h>

#include "task.h"
#include "sp4cuda.cpp"
#include "matrix4cuda.cpp"

#ifndef M_PI
#define M_PI (4.0*atan(1.0))
#endif

struct Info4cuda
{
	bool			valid;
	float			Color_r;
	float			Color_g;
	float			Color_b;
	bool			isEnter;	// ���荞��
	float			Distance;	// ��_�܂ł̋���
	float			Cross_x;	// ��_���Wx
	float			Cross_y;	// ��_���Wy
	float			Cross_z;	// ��_���Wz
	float			Vertical_x;	// �@���x�N�g��x
	float			Vertical_y;	// �@���x�N�g��y
	float			Vertical_z;	// �@���x�N�g��z
	float			Refractive;	// ���ܗ�
	NodeInfo		nodeInfo;	//
};

#define STACK_SIZE	5

struct StackData {
	bool			valid;
	float			Color_r;
	float			Color_g;
	float			Color_b;
	bool			isEnter;	// ���荞��
	float			Distance;	// ��_�܂ł̋���
	float			Cross_x;	// ��_���Wx
	float			Cross_y;	// ��_���Wy
	float			Cross_z;	// ��_���Wz
	float			Vertical_x;	// �@���x�N�g��x
	float			Vertical_y;	// �@���x�N�g��y
	float			Vertical_z;	// �@���x�N�g��z
	float			Refractive;	// ���ܗ�
	NodeInfo		nodeInfo;	//
};

struct Stack {
	int			Index;
	StackData	data[5];
};

#include "Sphere4cuda.cu"
//#include "Cone4cuda.cu"
//#include "Cylinder4cuda.cu"
//#include "Torus4cuda.cu"
//#include "Cube4cuda.cu"
#include "Plus4cuda.cu"

extern "C" __global__ 
void kernel(unsigned long* dst, int imageW, int imageH, matrix4cuda* pMatrix, sp4cuda* pLight, int taskIndex, Task* task)
{
    const int px = blockDim.x * blockIdx.x + threadIdx.x;
    const int py = blockDim.y * blockIdx.y + threadIdx.y;

	if (px >= imageW || py >= imageH)
		return;

	dst[px + py * imageW] = px + py * imageW;
	
	float rx = 20.0 * px / imageW - 10.0;
	float ry = 20.0 * py / imageH - 10.0;

	sp4cuda K = sp_sp(0.01 * rx / 20.0, 0.01 * ry / 20.0, 0.01);
	sp4cuda L = sp_sp(rx, ry, -20);


	K = sp_sp3(matrix_minus(matrix_multiple(*pMatrix, matrix_matrix3(sp_plus(K,L))), matrix_multiple(*pMatrix, matrix_matrix3(L))));
	L = sp_sp3(matrix_multiple(*pMatrix, matrix_matrix3(L)));
	
	sp4cuda Light = *pLight;
	sp4cuda ans = sp_sp(0,0,128);

	enum eMode {
		eNone, eReflect, eThrough
	};
	eMode mode = eNone;

	Info4cuda info;
	info.valid = false;
	
	for (int i = 0; i < 1; i++) {
		
		switch (mode) {
		case eNone:
			break;
		case eReflect:
			// ���˗��ŐF��������B
			{
			sp4cuda s = sp_plus(
				sp_multiple2(info.nodeInfo.m_Reflect, ans),
				sp_multiple2(1 - info.nodeInfo.m_Reflect, sp_sp(info.Color_r, info.Color_g, info.Color_b)));
			info.Color_r = s.x;
			info.Color_g = s.y;
			info.Color_b = s.z;
			}
			break;
		case eThrough:
			// ���ߗ��ŐF��������B
			{
			sp4cuda s = sp_plus(
				sp_multiple2(info.nodeInfo.m_Through, ans),
				sp_multiple2(1 - info.nodeInfo.m_Through, sp_sp(info.Color_r, info.Color_g, info.Color_b)));
			info.Color_r = s.x;
			info.Color_g = s.y;
			info.Color_b = s.z;
			}
			break;
		}

		Stack stack;
		stack.Index = 0;
		for (int idx = 0; idx < taskIndex; idx++) {
			matrix4cuda m = matrix_matrix(4,4,task[idx].m);
			sp4cuda L2 = sp_sp3(matrix_multiple(m, matrix_matrix3(L)));
			sp4cuda K2 = sp_minus(sp_sp3(matrix_multiple(m, matrix_matrix3(sp_plus(K,L)))), L2);

			Info4cuda	inf;
			switch (task[idx].type) {
			case SPHERE:
				inf = GetInfo_Sphere(task[idx], K2, L2);
				break;
			case PLANE:
				break;
			case PLUS:
				inf = GetInfo_Plus(task[idx], K2, L2, &stack);
				stack.Index -= task[idx].data.gathering.member;
				break;
			case MINUS:
				break;
			case MULTIPLE:
				break;
			case CONE:
				//inf = GetInfo_Cone(task[idx], K2, L2);
				break;
			case CYLINDER:
				//inf = GetInfo_Cylinder(task[idx], K2, L2);
				break;
			case TORUS:
				//inf = GetInfo_Torus(task[idx], K2, L2);
				break;
			case POLYGON:
				break;
			case CUBE:
				//inf = GetInfo_Cube(task[idx], K2, L2);
				break;
			case TEAPOT:
				break;
			default:
				break;
			}

			sp4cuda vertical = sp_sp(inf.Vertical_x, inf.Vertical_y, inf.Vertical_z);
			sp4cuda cross = sp_sp(inf.Cross_x, inf.Cross_y, inf.Cross_z);
			matrix4cuda Inv_m = matrix_Inv(m);
			sp4cuda vc = sp_plus(vertical, cross);
			matrix4cuda ivc = matrix_multiple(Inv_m, matrix_matrix3(vc));
			vertical = sp_sp3(matrix_minus(matrix_multiple(Inv_m, matrix_matrix3(vc)),
							matrix_multiple(Inv_m, matrix_matrix3(cross))));
			cross = sp_sp3(matrix_multiple(Inv_m, matrix_matrix3(cross)));
			sp4cuda cL = sp_minus(cross, L);
			float distance = sp_abs(cL);
			float refractive = inf.nodeInfo.m_Refractive;
			if (inf.isEnter)
				refractive = 1 / refractive;
			StackData*	sd = &(stack.data[stack.Index]);
			// inf ���X�^�b�N�ɐςށB
			sd->valid		= inf.valid;
			sd->Color_r		= inf.Color_r;
			sd->Color_g		= inf.Color_g;
			sd->Color_b		= inf.Color_b;
			sd->isEnter		= inf.isEnter;	// ���荞��
			sd->Distance	= distance;		// ��_�܂ł̋���
			sd->Cross_x		= cross.x;		// ��_���Wx
			sd->Cross_y		= cross.y;		// ��_���Wy
			sd->Cross_z		= cross.z;		// ��_���Wz
			sd->Vertical_x	= vertical.x;	// �@���x�N�g��x
			sd->Vertical_y	= vertical.y;	// �@���x�N�g��y
			sd->Vertical_z	= vertical.z;	// �@���x�N�g��z
			sd->Refractive	= refractive;	// ���ܗ�
			sd->nodeInfo	= inf.nodeInfo;	//
			stack.Index++;
			info = inf;
			return;
// OK return;
		}
// NG return;
		if (!info.valid)
			break;

		sp4cuda k = sp_e(K);
		sp4cuda v = sp_sp(info.Vertical_x, info.Vertical_y, info.Vertical_y);

		v = sp_e(v);
		
/*
		// ���˗�������ꍇ�A
		if (info.nodeInfo.m_Reflect > 0) {
			sp k2 = k - 2 * (v * k) * v;
			sp cross(info.Cross_x, info.Cross_y, info.Cross_z);
			sp l2 = cross + 1E-05 * k2;
			// ���˂��������x�N�g������F���擾�B
			K = k2, L = l2, mode = eReflect;
			continue;
		}

		// ���ߗ�������ꍇ�A
		if (info.nodeInfo.m_Through > 0) {
			float r = info.Refractive;
			float i = k * v;
			sp k2 = r * (k -i * v - sqrtf(r * r - 1.0 + i * i) * v);
			sp cross(info.Cross_x, info.Cross_y, info.Cross_z);
			sp l2 = cross + 1E-05 * k2;
			// ���܂��������x�N�g������F���擾�B
			K = k2, L = l2, mode = eThrough;
			continue;
		}
*/
		// �������F��␳�B
		sp4cuda vertical = sp_sp(info.Vertical_x, info.Vertical_y, info.Vertical_z);
		
		float	x = -sp_internal_multiple(sp_e(Light), sp_e(vertical));
		x = (x > 0.0) ? x : 0.0;
		float t = 64 + 191 * sinf(M_PI / 2 * x);
		float b = 191 * (1 - cosf(M_PI / 2 * x));
		
		ans = sp_plus(sp_divide(sp_multiple2(t - b, sp_sp(info.Color_r, info.Color_g, info.Color_b)), 255), sp_sp(b,b,b));
	}

	dst[px + py * imageW] = RGB(ans.x, ans.y, ans.z);
}

// Increase the grid size by 1 if the image width or height does not divide evenly
// by the thread block dimensions
inline int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
} // iDivUp

extern "C"
void DoCuda(unsigned long* out, int imageW, int imageH, matrix4cuda m, sp4cuda light, int taskIndex, Task* task)
{
	matrix4cuda*	dMatrix;
	sp4cuda*		dLight;
	Task*			dTask;
	unsigned long*	d_data;
//	Stack*			dStack;
    unsigned int mem_size = imageW * imageH * sizeof(unsigned long);
 
    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, mem_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&dMatrix, sizeof(matrix4cuda)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dLight, sizeof(sp4cuda)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dTask, taskIndex * sizeof(Task)));
//	CUDA_SAFE_CALL(hipMalloc((void**)&dStack, sizeof(Stack)));

	CUDA_SAFE_CALL(hipMemcpy(d_data, out, mem_size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dMatrix, &m, sizeof(matrix4cuda), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dLight, &light, sizeof(sp4cuda), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dTask, task, taskIndex * sizeof(Task), hipMemcpyHostToDevice));

    dim3 threads(16,16);
    dim3 grid(iDivUp(imageW, 16), iDivUp(imageH, 16));

	// execute the kernel
	kernel<<< grid, threads >>>(d_data, imageW, imageH, dMatrix, dLight, taskIndex, dTask);
	CUT_CHECK_ERROR("kernel failed.");

    // copy results from device to host
    CUDA_SAFE_CALL(hipMemcpy(out, d_data, mem_size, hipMemcpyDeviceToHost));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_data));
    CUDA_SAFE_CALL(hipFree(dMatrix));
    CUDA_SAFE_CALL(hipFree(dLight));
    CUDA_SAFE_CALL(hipFree(dTask));
//    CUDA_SAFE_CALL(hipFree(dStack));
}


#if 0
// The dimensions of the thread block
#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16

#define ABS(n) ((n) < 0 ? -(n) : (n))

// The Mandelbrot CUDA GPU thread function
template<class T>
__global__ void Mandelbrot0(unsigned long* dst, const int imageW, const int imageH, const int crunch, const T xOff, const T yOff, const T scale, const uchar4 colors, const int frame, const int animationFrame, matrix4cuda* pMatrix, sp4cuda* pLight)
{
    const int px = blockDim.x * blockIdx.x + threadIdx.x;
    const int py = blockDim.y * blockIdx.y + threadIdx.y;

	if (px < imageW && py < imageH) {
   		float rx = 20.0 * px / imageW - 10.0;
		float ry = 20.0 * py / imageH - 10.0;

		sp4cuda k = sp_sp(0.01 * rx / 20.0, 0.01 * ry / 20.0, 0.01);
		sp4cuda l = sp_sp(rx, ry, -20);
	    
		k = sp_sp(matrix_minus(matrix_multiple(*pMatrix, matrix_matrix(sp_plus(k, l))), matrix_multiple(*pMatrix, matrix_matrix(l))));
		l = sp_sp(matrix_multiple(*pMatrix, matrix_matrix(l)));

		sp4cuda c = GetColor(k, l, *pLight);

		dst[px + py * imageW] = RGB(c.x, c.y, c.z);
	}
} // Mandelbrot0


// The host CPU Mandebrot thread spawner
void RunMandelbrot0(unsigned long* out, const int imageW, const int imageH, const int crunch, const float xOff, const float yOff, const float scale, const uchar4 colors, const int frame, const int animationFrame, const matrix4cuda* m, const sp4cuda* light)
{
	matrix4cuda*		pMatrix;
	sp4cuda*			pLight;
	unsigned long* d_data;
    unsigned int mem_size = imageW * imageH * sizeof(unsigned long);
    
    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, mem_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&pMatrix, sizeof(matrix4cuda)));
	CUDA_SAFE_CALL(hipMalloc((void**)&pLight, sizeof(sp4cuda)));

	CUDA_SAFE_CALL(hipMemcpy(pMatrix, m, sizeof(matrix4cuda), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(pLight, light, sizeof(sp4cuda), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol<int>(cTaskIndex, &sTaskIndex, sizeof(int)));

    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

	Mandelbrot0<float><<<grid, threads>>>(out, imageW, imageH, crunch, (float)xOff, (float)yOff, (float)scale, colors, frame, animationFrame, pMatrix, pLight);
    CUT_CHECK_ERROR("Mandelbrot kernel execution failed.\n");

    // copy results from device to host
    CUDA_SAFE_CALL(hipMemcpy(out, d_data, mem_size, hipMemcpyDeviceToHost));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_data));
    CUDA_SAFE_CALL(hipFree(pMatrix));
    CUDA_SAFE_CALL(hipFree(pLight));
} // RunMandelbrot0

#endif