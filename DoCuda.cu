#include "hip/hip_runtime.h"
#include <cutil.h>
#include <hip/hip_runtime.h>
#include <d3dx9.h>
#include <atltypes.h>

#define TARGET __device__
#include "matrix.h"
#include "info.h"

#ifndef M_PI
#define M_PI (4.0*atan(1.0))
#endif

#include "BaseNode.h"
#include "BaseNode.cpp"


__device__ matrix		Matrix;
__device__ BaseNode*	Root;
__device__ sp			Light;

// �����x�N�g��(Kt+L)����F��Ԃ��B
TARGET sp BaseNode::GetColor(const sp* K, const sp* L, int nest)
{
	Info	info;

	// �ċA�����P�O���z���閔�́A��_�����݂��Ȃ��ꍇ�A
	if (nest > 10 || !GetInfo2(K, L, &info))
		return sp(127, 127, 127);

	sp k = K->e();
	sp v = info.Vertical.e();

	// ���˗�������ꍇ�A
	if (info.pNode->m_Reflect > 0) {
		sp k2 = k - 2 * (v * k) * v;
		sp l2 = info.Cross + 1E-05 * k2;
		// ���˂��������x�N�g������F���擾�B
		sp c = Root->GetColor(&k2, &l2, nest + 1);
		// ���˗��ŐF��������B
		info.Material = (info.pNode->m_Reflect * c + (1 - info.pNode->m_Reflect) * sp(info.Material)).getMaterial();
	}

	// ���ߗ�������ꍇ�A
	if (info.pNode->m_Through > 0) {
		double r = info.Refractive;
		double i = k * v;
		sp k2 = r * (k -i * v - sqrt(r * r - 1.0 + i * i) * v);
		sp l2 = info.Cross + 1E-05 * k2;
		// ���܂��������x�N�g������F���擾�B
		sp c = Root->GetColor(&k2, &l2, nest + 1);
		// ���ߗ��ŐF��������B
		info.Material = (info.pNode->m_Through * c + (1 - info.pNode->m_Through) * sp(info.Material)).getMaterial();
	}

	// �������F��␳�B
	double	x = -Light.e() * info.Vertical.e();
	x = (x > 0.0) ? x : 0.0;
	double t = 64 + 191 * sin(M_PI / 2 * x);
	double b = 191 * (1 - cos(M_PI / 2 * x));

	return (t - b) * sp(info.Material) / 255 + sp(b,b,b);
}

__global__ void kernel(unsigned long* dst, const int imageW, const int imageH)
{
    const int px = blockDim.x * blockIdx.x + threadIdx.x;
    const int py = blockDim.y * blockIdx.y + threadIdx.y;
   	double rx = 20.0 * px / imageW - 10.0;
	double ry = 20.0 * py / imageH - 10.0;

	sp k(0.01 * rx / 20.0, 0.01 * ry / 20.0, 0.01);
	sp l(rx, ry, -20);
    
	k = Matrix * (k + l) - Matrix * l;
	l = Matrix * l;

	sp c = Root->GetColor(&k, &l, 0);
	
	if (px <= imageW && py <= imageH) {
		dst[px + py * imageW] = RGB(c.x, c.y, c.z);
	}
}

void DoCuda(unsigned long* out, class BaseNode* root, const int imageW, const int imageH, const matrix* m, const sp* light)
{
	hipError_t err;

	unsigned long* d_data;
    unsigned int mem_size = imageW * imageH * sizeof(unsigned long);
	if (hipSuccess != (err = hipMalloc((void**)&d_data, mem_size))) {
		MessageBox(0, hipGetErrorString(err), "hipMalloc:1", MB_OK);
		RaiseException(0,0,0,0);
		return;
	}

    if (hipSuccess != (err = hipMemcpy(&Root, &root, sizeof(BaseNode*), hipMemcpyHostToDevice))) {
		MessageBox(0, hipGetErrorString(err), "hipMemcpy:1", MB_OK);
		RaiseException(0,0,0,0);
	}

    if (hipSuccess != (err = hipMemcpy(&Matrix, m, sizeof(matrix), hipMemcpyHostToDevice))) {
		MessageBox(0, hipGetErrorString(err), "hipMemcpy:1", MB_OK);
		RaiseException(0,0,0,0);
	}

    if (hipSuccess != (err = hipMemcpy(&Light, light, sizeof(sp), hipMemcpyHostToDevice))) {
		MessageBox(0, hipGetErrorString(err), "hipMemcpy:2", MB_OK);
		RaiseException(0,0,0,0);
	}
	
    dim3 threads(16,16);
    dim3 grid(16,16);
 
	// execute the kernel
	kernel<<< grid, threads >>>(d_data, imageW, imageH);

    if (hipSuccess != hipGetLastError()) {
		MessageBox(0, hipGetErrorString(err), "hipGetLastError", MB_OK);
		RaiseException(0,0,0,0);
		return;
	}
	
    if (hipSuccess != (err = hipDeviceSynchronize())) {
		MessageBox(0, hipGetErrorString(err), "hipDeviceSynchronize", MB_OK);
		RaiseException(0,0,0,0);
    	return;
    }

    // copy results from device to host
    if (hipSuccess != (err = hipMemcpy(out, d_data, mem_size, hipMemcpyDeviceToHost))) {
		MessageBox(0, hipGetErrorString(err), "hipMemcpy:2", MB_OK);
		RaiseException(0,0,0,0);
	}

    // cleanup memory
    if (hipSuccess != (err = hipFree(d_data))) {
		MessageBox(0, hipGetErrorString(err), "hipFree", MB_OK);
		RaiseException(0,0,0,0);
		return;
	}
}

