
#include <hip/hip_runtime.h>
// RayTraceView.cpp : CRayTraceView �N���X�̓���̒�`���s���܂��B
//

// The dimensions of the thread block
#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16

// Increase the grid size by 1 if the image width or height does not divide evenly
// by the thread block dimensions
inline int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
} // iDivUp

void DoCuda(const int imageW, const int imageH)
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));
}

