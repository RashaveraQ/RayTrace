#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cutil4win.h"
#include <d3dx9.h>
#include <atltypes.h>

#define TARGET __device__
#include "task.h"
#include "matrix.h"
//#include "info.h"

__constant__ Task cTask[100];
__constant__ int  cTaskIndex;
static int sTaskIndex = 0;

__global__ void kernel(unsigned long* dst, int imageW, int imageH, matrix* pMatrix, sp* pLight)
{
    const int px = blockDim.x * blockIdx.x + threadIdx.x;
    const int py = blockDim.y * blockIdx.y + threadIdx.y;
   	double rx = 20.0 * px / imageW - 10.0;
	double ry = 20.0 * py / imageH - 10.0;

	sp k(0.01 * rx / 20.0, 0.01 * ry / 20.0, 0.01);
	sp l(rx, ry, -20);
    
	k = *pMatrix * (k + l) - *pMatrix * l;
	l = *pMatrix * l;

//	sp c = Root->GetColor(&k, &l, 0);
	sp c((imageW * cTaskIndex) % 256, (imageH * cTaskIndex) % 256, 0);
	if (px <= imageW && py <= imageH) {
		dst[px + py * imageW] = RGB(c.x, c.y, c.z);
	}
}

void DoCuda(unsigned long* out, const int imageW, const int imageH, const matrix* m, const sp* light)
{
	matrix*		pMatrix;
	sp*			pLight;
	unsigned long* d_data;
    unsigned int mem_size = imageW * imageH * sizeof(unsigned long);
    
    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, mem_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&pMatrix, sizeof(matrix)));
	CUDA_SAFE_CALL(hipMalloc((void**)&pLight, sizeof(sp)));

	CUDA_SAFE_CALL(hipMemcpy(pMatrix, m, sizeof(matrix), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(pLight, light, sizeof(sp), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(&cTaskIndex), &sTaskIndex, sizeof(int)));

    dim3 threads(16,16);
    dim3 grid(16,16);
//    dim3 grid(iDivUp(imageW, 16), iDivUp(imageH, 16));

	// execute the kernel
	kernel<<< grid, threads >>>(d_data, imageW, imageH, pMatrix, pLight);
	CUT_CHECK_ERROR("kernel failed.");

    // copy results from device to host
    CUDA_SAFE_CALL(hipMemcpy(out, d_data, mem_size, hipMemcpyDeviceToHost));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_data));
    CUDA_SAFE_CALL(hipFree(pMatrix));
    CUDA_SAFE_CALL(hipFree(pLight));
}

void ClearTask()
{
	sTaskIndex = 0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(&cTaskIndex), &sTaskIndex, sizeof(int)));
}

void AddTask(const Task& task)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cTask + sTaskIndex), &task, sizeof(Task)));
	sTaskIndex++;
}
