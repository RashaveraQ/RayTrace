#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cutil4win.h"
#include <d3dx9.h>
#include <atltypes.h>

#define TARGET __device__
#include "task.h"
#include "sp.h"
#include "matrix.h"

#ifndef M_PI
#define M_PI (4.0*atan(1.0))
#endif

__constant__ Task cTask[100];
__constant__ int  cTaskIndex;
static int sTaskIndex = 0;

struct Info4cuda
{
	bool			valid;
	D3DMATERIAL9	Material;
	BOOL			isEnter;	// ���荞��
	double			Distance;	// ��_�܂ł̋���
	double			Cross_x;	// ��_���Wx
	double			Cross_y;	// ��_���Wy
	double			Cross_z;	// ��_���Wz
	double			Vertical_x;	// �@���x�N�g��x
	double			Vertical_y;	// �@���x�N�g��y
	double			Vertical_z;	// �@���x�N�g��z
	double			Refractive;	// ���ܗ�
	NodeInfo		nodeInfo;	//
};

#define STACK_SIZE	20

struct Stack {
	int				Index;
	bool			valid		[STACK_SIZE];
	D3DMATERIAL9	Material	[STACK_SIZE];
	BOOL			isEnter		[STACK_SIZE];	// ���荞��
	double			Distance	[STACK_SIZE];	// ��_�܂ł̋���
	double			Cross_x		[STACK_SIZE];	// ��_���Wx
	double			Cross_y		[STACK_SIZE];	// ��_���Wy
	double			Cross_z		[STACK_SIZE];	// ��_���Wz
	double			Vertical_x	[STACK_SIZE];	// �@���x�N�g��x
	double			Vertical_y	[STACK_SIZE];	// �@���x�N�g��y
	double			Vertical_z	[STACK_SIZE];	// �@���x�N�g��z
	double			Refractive	[STACK_SIZE];	// ���ܗ�
	NodeInfo		nodeInfo	[STACK_SIZE];	//
};

#include "Sphere4cuda.cu"
#include "Cone4cuda.cu"
#include "Cylinder4cuda.cu"
#include "Torus4cuda.cu"
#include "Cube4cuda.cu"
#include "Plus4cuda.cu"

__device__
bool GetInfo2(const sp& K, const sp& L, Info4cuda& info)
{
	Stack stack;
	stack.Index = 0;
	
	for (int idx = 0; idx < cTaskIndex; idx++) {
		Info4cuda	inf;
		matrix m;
		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				m.m_data[i][j] = cTask[idx].m[i][j];
			}
		}

		sp L2 = m * L;
		sp K2 = m * (K + L) - L2;

		switch (cTask[idx].type) {
		case SPHERE:
			GetInfo_Sphere(cTask[idx], K2, L2, inf);
			break;
		case PLANE:
			break;
		case PLUS:
			GetInfo_Plus(cTask[idx], K2, L2, inf, stack);
			break;
		case MINUS:
			break;
		case MULTIPLE:
			break;
		case CONE:
			GetInfo_Cone(cTask[idx], K2, L2, inf);
			break;
		case CYLINDER:
			GetInfo_Cylinder(cTask[idx], K2, L2, inf);
			break;
		case TORUS:
			GetInfo_Torus(cTask[idx], K2, L2, inf);
			break;
		case POLYGON:
			break;
		case CUBE:
			GetInfo_Cube(cTask[idx], K2, L2, inf);
			break;
		case TEAPOT:
			break;
		default:
			break;
		}
		
		sp vertical(inf.Vertical_x, inf.Vertical_y, inf.Vertical_z);
		sp cross(inf.Cross_x, inf.Cross_y, inf.Cross_z);
		
		matrix Inv_m = m.Inv();
		vertical = Inv_m * (vertical + cross) - Inv_m * cross;
		cross = Inv_m * cross;
		inf.Distance = (cross - L).abs();
		inf.Refractive = inf.nodeInfo.m_Refractive;
		if (inf.isEnter)
			inf.Refractive = 1 / inf.Refractive;

		inf.Vertical_x = vertical.x;
		inf.Vertical_y = vertical.y;
		inf.Vertical_z = vertical.z;
		inf.Cross_x = cross.x;
		inf.Cross_y = cross.y;
		inf.Cross_z = cross.z;
		
		// inf ���X�^�b�N�ɐςށB
		stack.valid			[stack.Index] = inf.valid;
		stack.Material		[stack.Index] = inf.Material;
		stack.isEnter		[stack.Index] = inf.isEnter;	// ���荞��
		stack.Distance		[stack.Index] = inf.Distance;	// ��_�܂ł̋���
		stack.Cross_x		[stack.Index] = inf.Cross_x;	// ��_���Wx
		stack.Cross_y		[stack.Index] = inf.Cross_y;	// ��_���Wy
		stack.Cross_z		[stack.Index] = inf.Cross_z;	// ��_���Wz
		stack.Vertical_x	[stack.Index] = inf.Vertical_x;	// �@���x�N�g��x
		stack.Vertical_y	[stack.Index] = inf.Vertical_y;	// �@���x�N�g��y
		stack.Vertical_z	[stack.Index] = inf.Vertical_z;	// �@���x�N�g��z
		stack.Refractive	[stack.Index] = inf.Refractive;	// ���ܗ�
		stack.nodeInfo		[stack.Index] = inf.nodeInfo;	//
		stack.Index++;
		info = inf;
	}

	return info.valid;
}

__device__
sp GetColor(sp K, sp L, const sp& Light)
{
	sp ans(127,127,127);

	enum eMode {
		eNone, eReflect, eThrough
	};
	eMode mode = eNone;

	Info4cuda info;
	
	for (int i = 0; i < 10; i++) {
		
		switch (mode) {
		case eNone:
			break;
		case eReflect:
			// ���˗��ŐF��������B
			info.Material = (info.nodeInfo.m_Reflect * ans + (1 - info.nodeInfo.m_Reflect) * sp(info.Material)).getMaterial();
			break;
		case eThrough:
			// ���ߗ��ŐF��������B
			info.Material = (info.nodeInfo.m_Through * ans + (1 - info.nodeInfo.m_Through) * sp(info.Material)).getMaterial();
			break;
		}

		if (!GetInfo2(K, L, info))
			break;
			
		sp k = K.e();
		sp v(info.Vertical_x, info.Vertical_y, info.Vertical_y);

		v = v.e();
/*
		// ���˗�������ꍇ�A
		if (info.nodeInfo.m_Reflect > 0) {
			sp k2 = k - 2 * (v * k) * v;
			sp cross(info.Cross_x, info.Cross_y, info.Cross_z);
			sp l2 = cross + 1E-05 * k2;
			// ���˂��������x�N�g������F���擾�B
			K = k2, L = l2, mode = eReflect;
			continue;
		}

		// ���ߗ�������ꍇ�A
		if (info.nodeInfo.m_Through > 0) {
			double r = info.Refractive;
			double i = k * v;
			sp k2 = r * (k -i * v - sqrt(r * r - 1.0 + i * i) * v);
			sp cross(info.Cross_x, info.Cross_y, info.Cross_z);
			sp l2 = cross + 1E-05 * k2;
			// ���܂��������x�N�g������F���擾�B
			K = k2, L = l2, mode = eThrough;
			continue;
		}
*/
		// �������F��␳�B
		sp vertical(info.Vertical_x, info.Vertical_y, info.Vertical_z);
		
		double	x = -Light.e() * vertical.e();
		x = (x > 0.0) ? x : 0.0;
		double t = 64 + 191 * sin(M_PI / 2 * x);
		double b = 191 * (1 - cos(M_PI / 2 * x));
		
		ans =  (t - b) * sp(info.Material) / 255 + sp(b,b,b);
	}

	return ans;
}

__global__
void kernel(unsigned long* dst, int imageW, int imageH, matrix* pMatrix, sp* pLight)
{
    const int px = blockDim.x * blockIdx.x + threadIdx.x;
    const int py = blockDim.y * blockIdx.y + threadIdx.y;
   	double rx = 20.0 * px / imageW - 10.0;
	double ry = 20.0 * py / imageH - 10.0;

	sp k(0.01 * rx / 20.0, 0.01 * ry / 20.0, 0.01);
	sp l(rx, ry, -20);
    
	k = *pMatrix * (k + l) - *pMatrix * l;
	l = *pMatrix * l;

	sp c = GetColor(k, l, *pLight);

	if (px <= imageW && py <= imageH) {
		dst[px + py * imageW] = RGB(c.x, c.y, c.z);
	}
}

void DoCuda(unsigned long* out, const int imageW, const int imageH, const matrix* m, const sp* light)
{
	matrix*		pMatrix;
	sp*			pLight;
	unsigned long* d_data;
    unsigned int mem_size = imageW * imageH * sizeof(unsigned long);
    

    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, mem_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&pMatrix, sizeof(matrix)));
	CUDA_SAFE_CALL(hipMalloc((void**)&pLight, sizeof(sp)));

	CUDA_SAFE_CALL(hipMemcpy(pMatrix, m, sizeof(matrix), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(pLight, light, sizeof(sp), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol<int>(cTaskIndex, &sTaskIndex, sizeof(int)));

    dim3 threads(1,1);
    dim3 grid(488,488);
//    dim3 grid(iDivUp(imageW, 16), iDivUp(imageH, 16));

	// execute the kernel
	kernel<<< grid, threads >>>(d_data, imageW, imageH, pMatrix, pLight);
	CUT_CHECK_ERROR("kernel failed.");

    // copy results from device to host
    CUDA_SAFE_CALL(hipMemcpy(out, d_data, mem_size, hipMemcpyDeviceToHost));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_data));
    CUDA_SAFE_CALL(hipFree(pMatrix));
    CUDA_SAFE_CALL(hipFree(pLight));
}

void ClearTask()
{
	sTaskIndex = 0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol<int>(cTaskIndex, &sTaskIndex, sizeof(int)));
}

void AddTask(const Task& task)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol<Task>(cTask[0], &task, sizeof(Task), sTaskIndex * sizeof(Task)));
	sTaskIndex++;
}
