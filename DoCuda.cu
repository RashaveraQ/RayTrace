#include "hip/hip_runtime.h"
#include <cutil.h>
#include <hip/hip_runtime.h>

#include <d3dx9.h>
#include <atltypes.h>

#define TARGET __device__
#include "NodeBase.h"
#include "NodeBase_kernel.cpp"

// The dimensions of the thread block
#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16

__global__ void kernel(unsigned long* dst, NodeBase* root, const sp& light, const int imageW, const int imageH, const matrix* m)
{
    const int px = blockDim.x * blockIdx.x + threadIdx.x;
    const int py = blockDim.y * blockIdx.y + threadIdx.y;
    
   	double rx = 20.0 * px / imageW - 10.0;
	double ry = 20.0 * py / imageH - 10.0;

	sp k(0.01 * rx / 20.0, 0.01 * ry / 20.0, 0.01);
	sp l(rx, ry, -20);

	k = *m * (k + l) - *m * l;
	l = *m * l;
 
	sp c = root->GetColor(root, light, k, l, 0);
	
	dst[px + py * imageW] = RGB(c.x, c.y, c.z);
}

// Increase the grid size by 1 if the image width or height does not divide evenly
// by the thread block dimensions
inline int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
} // iDivUp

void DoCuda(unsigned long* out, NodeBase* root, const sp& light, const int imageW, const int imageH, const matrix* m)
{
	unsigned long* d_data;
    const unsigned int mem_size = imageW * imageH * sizeof(unsigned long);

	CUDA_SAFE_CALL(hipMalloc((void**)&d_data, mem_size));
	
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));
 
	// execute the kernel
	kernel<<< grid, threads >>>(d_data, root, light, imageW, imageH, m);

	// check if kernel execution generated and error
	CUT_CHECK_ERROR("Kernel execution failed");

    // copy results from device to host
    CUDA_SAFE_CALL(hipMemcpy(out, d_data, mem_size, hipMemcpyDeviceToHost));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_data));
}

