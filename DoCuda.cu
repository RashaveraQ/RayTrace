#include "hip/hip_runtime.h"
#include <cutil.h>
#include <hip/hip_runtime.h>
#include <d3dx9.h>
#include <atltypes.h>
#include "sp.h"

// The dimensions of the thread block
#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16

__global__ void kernel(unsigned long* dst, void* root, const int imageW, const int imageH)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
}

// Increase the grid size by 1 if the image width or height does not divide evenly
// by the thread block dimensions
inline int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
} // iDivUp

void DoCuda(unsigned long* out, class Node* root, const int imageW, const int imageH)
{
	unsigned long* d_data;
    const unsigned int mem_size = imageW * imageH * sizeof(unsigned long);

	CUDA_SAFE_CALL(hipMalloc((void**)&d_data, mem_size));
	
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));
 
	// execute the kernel
	kernel<<< grid, threads >>>(d_data, (void*)root, imageW, imageH);

	// check if kernel execution generated and error
	CUT_CHECK_ERROR("Kernel execution failed");

    // copy results from device to host
    CUDA_SAFE_CALL(hipMemcpy(out, d_data, mem_size, hipMemcpyDeviceToHost));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_data));
}

