#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cutil4win.h"
//#include <atltypes.h>

#define TARGET __device__ extern "C"
#include "task.h"
#include "sp4cuda.cpp"
#include "matrix4cuda.cpp"

#ifndef M_PI
#define M_PI (4.0*atan(1.0))
#endif

//__constant__ Task cTask[100];
//__constant__ int  cTaskIndex;
static int sTaskIndex = 0;
static Task sTask[100];

struct Info4cuda
{
	bool			valid;
//	D3DMATERIAL9	Material;
	double			Color_r;
	double			Color_g;
	double			Color_b;
	bool			isEnter;	// ���荞��
	double			Distance;	// ��_�܂ł̋���
	double			Cross_x;	// ��_���Wx
	double			Cross_y;	// ��_���Wy
	double			Cross_z;	// ��_���Wz
	double			Vertical_x;	// �@���x�N�g��x
	double			Vertical_y;	// �@���x�N�g��y
	double			Vertical_z;	// �@���x�N�g��z
	double			Refractive;	// ���ܗ�
	NodeInfo		nodeInfo;	//
};

#define STACK_SIZE	20

struct Stack {
	int				Index;
	bool			valid		[STACK_SIZE];
//	D3DMATERIAL9	Material	[STACK_SIZE];
	double			Color_r		[STACK_SIZE];
	double			Color_g		[STACK_SIZE];
	double			Color_b		[STACK_SIZE];
	bool			isEnter		[STACK_SIZE];	// ���荞��
	double			Distance	[STACK_SIZE];	// ��_�܂ł̋���
	double			Cross_x		[STACK_SIZE];	// ��_���Wx
	double			Cross_y		[STACK_SIZE];	// ��_���Wy
	double			Cross_z		[STACK_SIZE];	// ��_���Wz
	double			Vertical_x	[STACK_SIZE];	// �@���x�N�g��x
	double			Vertical_y	[STACK_SIZE];	// �@���x�N�g��y
	double			Vertical_z	[STACK_SIZE];	// �@���x�N�g��z
	double			Refractive	[STACK_SIZE];	// ���ܗ�
	NodeInfo		nodeInfo	[STACK_SIZE];	//
};

#include "Sphere4cuda.cu"
//#include "Cone4cuda.cu"
//#include "Cylinder4cuda.cu"
//#include "Torus4cuda.cu"
//#include "Cube4cuda.cu"
#include "Plus4cuda.cu"

extern "C" __device__
bool GetInfo2(sp4cuda K, sp4cuda L, Info4cuda* info, int taskIndex, Task *task)
{
	Stack stack;
	stack.Index = 0;
	for (int idx = 0; idx < taskIndex; idx++) {
		Info4cuda	inf;
		matrix4cuda m = matrix_matrix(4,4,task[idx].m);

		sp4cuda L2 = sp_sp3(matrix_multiple(m, matrix_matrix3(L)));
		sp4cuda K2 = sp_minus(sp_sp3(matrix_multiple(m, matrix_matrix3(sp_plus(K,L)))), L2);

		switch (task[idx].type) {
		case SPHERE:
			GetInfo_Sphere(task[idx], K2, L2, &inf);
			break;
		case PLANE:
			break;
		case PLUS:
			GetInfo_Plus(task[idx], K2, L2, &inf, &stack);
			break;
		case MINUS:
			break;
		case MULTIPLE:
			break;
		case CONE:
			//GetInfo_Cone(task[idx], K2, L2, inf);
			break;
		case CYLINDER:
			//GetInfo_Cylinder(task[idx], K2, L2, inf);
			break;
		case TORUS:
			//GetInfo_Torus(task[idx], K2, L2, inf);
			break;
		case POLYGON:
			break;
		case CUBE:
			//GetInfo_Cube(task[idx], K2, L2, inf);
			break;
		case TEAPOT:
			break;
		default:
			break;
		}
		
		sp4cuda vertical = sp_sp(inf.Vertical_x, inf.Vertical_y, inf.Vertical_z);
		sp4cuda cross = sp_sp(inf.Cross_x, inf.Cross_y, inf.Cross_z);
		
		matrix4cuda Inv_m = matrix_Inv(m);
		vertical = sp_sp3(matrix_minus(matrix_multiple(Inv_m, matrix_matrix3(sp_plus(vertical, cross))),
						matrix_multiple(Inv_m, matrix_matrix3(cross))));
		cross = sp_sp3(matrix_multiple(Inv_m, matrix_matrix3(cross)));

		inf.Distance = sp_abs(sp_minus(cross, L));
		inf.Refractive = inf.nodeInfo.m_Refractive;
		if (inf.isEnter)
			inf.Refractive = 1 / inf.Refractive;

		inf.Vertical_x = vertical.x;
		inf.Vertical_y = vertical.y;
		inf.Vertical_z = vertical.z;
		inf.Cross_x = cross.x;
		inf.Cross_y = cross.y;
		inf.Cross_z = cross.z;
		
		// inf ���X�^�b�N�ɐςށB
		stack.valid			[stack.Index] = inf.valid;
		stack.Color_r		[stack.Index] = inf.Color_r;
		stack.Color_g		[stack.Index] = inf.Color_g;
		stack.Color_b		[stack.Index] = inf.Color_b;
		stack.isEnter		[stack.Index] = inf.isEnter;	// ���荞��
		stack.Distance		[stack.Index] = inf.Distance;	// ��_�܂ł̋���
		stack.Cross_x		[stack.Index] = inf.Cross_x;	// ��_���Wx
		stack.Cross_y		[stack.Index] = inf.Cross_y;	// ��_���Wy
		stack.Cross_z		[stack.Index] = inf.Cross_z;	// ��_���Wz
		stack.Vertical_x	[stack.Index] = inf.Vertical_x;	// �@���x�N�g��x
		stack.Vertical_y	[stack.Index] = inf.Vertical_y;	// �@���x�N�g��y
		stack.Vertical_z	[stack.Index] = inf.Vertical_z;	// �@���x�N�g��z
		stack.Refractive	[stack.Index] = inf.Refractive;	// ���ܗ�
		stack.nodeInfo		[stack.Index] = inf.nodeInfo;	//
		stack.Index++;
		*info = inf;
	}

	return info->valid;
}

extern "C" __device__ 
sp4cuda GetColor(sp4cuda K, sp4cuda L, sp4cuda Light, int taskIndex, Task* task)
{
	sp4cuda ans = sp_sp(127,127,127);

	enum eMode {
		eNone, eReflect, eThrough
	};
	eMode mode = eNone;

	Info4cuda info;
	
	for (int i = 0; i < 10; i++) {
		
		switch (mode) {
		case eNone:
			break;
		case eReflect:
			// ���˗��ŐF��������B
			{
			sp4cuda s = sp_plus(
				sp_multiple2(info.nodeInfo.m_Reflect, ans),
				sp_multiple2(1 - info.nodeInfo.m_Reflect, sp_sp(info.Color_r, info.Color_g, info.Color_b)));
			info.Color_r = s.x;
			info.Color_g = s.y;
			info.Color_b = s.z;
			}
			break;
		case eThrough:
			// ���ߗ��ŐF��������B
			{
			sp4cuda s = sp_plus(
				sp_multiple2(info.nodeInfo.m_Through, ans),
				sp_multiple2(1 - info.nodeInfo.m_Through, sp_sp(info.Color_r, info.Color_g, info.Color_b)));
			info.Color_r = s.x;
			info.Color_g = s.y;
			info.Color_b = s.z;
			}
			break;
		}

		if (!GetInfo2(K, L, &info, taskIndex, task))
			break;
			
		sp4cuda k = sp_e(K);
		sp4cuda v = sp_sp(info.Vertical_x, info.Vertical_y, info.Vertical_y);

		v = sp_e(v);
/*
		// ���˗�������ꍇ�A
		if (info.nodeInfo.m_Reflect > 0) {
			sp k2 = k - 2 * (v * k) * v;
			sp cross(info.Cross_x, info.Cross_y, info.Cross_z);
			sp l2 = cross + 1E-05 * k2;
			// ���˂��������x�N�g������F���擾�B
			K = k2, L = l2, mode = eReflect;
			continue;
		}

		// ���ߗ�������ꍇ�A
		if (info.nodeInfo.m_Through > 0) {
			double r = info.Refractive;
			double i = k * v;
			sp k2 = r * (k -i * v - sqrt(r * r - 1.0 + i * i) * v);
			sp cross(info.Cross_x, info.Cross_y, info.Cross_z);
			sp l2 = cross + 1E-05 * k2;
			// ���܂��������x�N�g������F���擾�B
			K = k2, L = l2, mode = eThrough;
			continue;
		}
*/
		// �������F��␳�B
		sp4cuda vertical = sp_sp(info.Vertical_x, info.Vertical_y, info.Vertical_z);
		
		double	x = -sp_internal_multiple(sp_e(Light), sp_e(vertical));
		x = (x > 0.0) ? x : 0.0;
		double t = 64 + 191 * sin(M_PI / 2 * x);
		double b = 191 * (1 - cos(M_PI / 2 * x));
		
		ans = sp_plus(sp_divide(sp_multiple2(t - b, sp_sp(info.Color_r, info.Color_g, info.Color_b)), 255), sp_sp(b,b,b));
	}

	return ans;
}

extern "C" __global__ 
void kernel(unsigned long* dst, int imageW, int imageH, matrix4cuda* pMatrix, sp4cuda* pLight, int taskIndex, Task* task)
{
    const int px = blockDim.x * blockIdx.x + threadIdx.x;
    const int py = blockDim.y * blockIdx.y + threadIdx.y;

	if (px < imageW && py < imageH) {
   		double rx = 20.0 * px / imageW - 10.0;
		double ry = 20.0 * py / imageH - 10.0;

		sp4cuda k = sp_sp(0.01 * rx / 20.0, 0.01 * ry / 20.0, 0.01);
		sp4cuda l = sp_sp(rx, ry, -20);
		k = sp_sp3(matrix_minus(matrix_multiple(*pMatrix, matrix_matrix3(sp_plus(k,l))), matrix_multiple(*pMatrix, matrix_matrix3(l))));
		l = sp_sp3(matrix_multiple(*pMatrix, matrix_matrix3(l)));

		sp4cuda c = GetColor(k, l, *pLight, taskIndex, task);

		dst[px + py * imageW] = RGB(c.x, c.y, c.z);
	}
}

// Increase the grid size by 1 if the image width or height does not divide evenly
// by the thread block dimensions
inline int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
} // iDivUp

extern "C"
void DoCuda(unsigned long* out, int imageW, int imageH, matrix4cuda m, sp4cuda light)
{
	matrix4cuda*		pMatrix;
	sp4cuda*			pLight;
	Task*				dTask;
	unsigned long* d_data;
    unsigned int mem_size = imageW * imageH * sizeof(unsigned long);
    
    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, mem_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&pMatrix, sizeof(matrix4cuda)));
	CUDA_SAFE_CALL(hipMalloc((void**)&pLight, sizeof(sp4cuda)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dTask, sTaskIndex * sizeof(Task)));

	CUDA_SAFE_CALL(hipMemcpy(pMatrix, &m, sizeof(matrix4cuda), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(pLight, &light, sizeof(sp4cuda), hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpyToSymbol<int>(cTaskIndex, &sTaskIndex, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(dTask, sTask, sTaskIndex * sizeof(Task), hipMemcpyHostToDevice));

    dim3 threads(16,16);
    dim3 grid(iDivUp(imageW, 16), iDivUp(imageH, 16));

	// execute the kernel
	kernel<<< grid, threads >>>(d_data, imageW, imageH, pMatrix, pLight, sTaskIndex, dTask);
	CUT_CHECK_ERROR("kernel failed.");

    // copy results from device to host
    CUDA_SAFE_CALL(hipMemcpy(out, d_data, mem_size, hipMemcpyDeviceToHost));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_data));
    CUDA_SAFE_CALL(hipFree(pMatrix));
    CUDA_SAFE_CALL(hipFree(pLight));
    CUDA_SAFE_CALL(hipFree(dTask));
}

extern "C"
void ClearTask()
{
	sTaskIndex = 0;
//	CUDA_SAFE_CALL(hipMemcpyToSymbol<int>(cTaskIndex, &sTaskIndex, sizeof(int)));
}

extern "C"
void AddTask(Task task)
{
//	CUDA_SAFE_CALL(hipMemcpyToSymbol<Task>(cTask[0], &task, sizeof(Task), sTaskIndex * sizeof(Task)));
	memcpy(sTask + sTaskIndex, &task, sizeof(Task));
	sTaskIndex++;
}

#if 0
// The dimensions of the thread block
#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16

#define ABS(n) ((n) < 0 ? -(n) : (n))

// The Mandelbrot CUDA GPU thread function
template<class T>
__global__ void Mandelbrot0(unsigned long* dst, const int imageW, const int imageH, const int crunch, const T xOff, const T yOff, const T scale, const uchar4 colors, const int frame, const int animationFrame, matrix4cuda* pMatrix, sp4cuda* pLight)
{
    const int px = blockDim.x * blockIdx.x + threadIdx.x;
    const int py = blockDim.y * blockIdx.y + threadIdx.y;

	if (px < imageW && py < imageH) {
   		double rx = 20.0 * px / imageW - 10.0;
		double ry = 20.0 * py / imageH - 10.0;

		sp4cuda k = sp_sp(0.01 * rx / 20.0, 0.01 * ry / 20.0, 0.01);
		sp4cuda l = sp_sp(rx, ry, -20);
	    
		k = sp_sp(matrix_minus(matrix_multiple(*pMatrix, matrix_matrix(sp_plus(k, l))), matrix_multiple(*pMatrix, matrix_matrix(l))));
		l = sp_sp(matrix_multiple(*pMatrix, matrix_matrix(l)));

		sp4cuda c = GetColor(k, l, *pLight);

		dst[px + py * imageW] = RGB(c.x, c.y, c.z);
	}
} // Mandelbrot0


// The host CPU Mandebrot thread spawner
void RunMandelbrot0(unsigned long* out, const int imageW, const int imageH, const int crunch, const double xOff, const double yOff, const double scale, const uchar4 colors, const int frame, const int animationFrame, const matrix4cuda* m, const sp4cuda* light)
{
	matrix4cuda*		pMatrix;
	sp4cuda*			pLight;
	unsigned long* d_data;
    unsigned int mem_size = imageW * imageH * sizeof(unsigned long);
    
    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, mem_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&pMatrix, sizeof(matrix4cuda)));
	CUDA_SAFE_CALL(hipMalloc((void**)&pLight, sizeof(sp4cuda)));

	CUDA_SAFE_CALL(hipMemcpy(pMatrix, m, sizeof(matrix4cuda), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(pLight, light, sizeof(sp4cuda), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol<int>(cTaskIndex, &sTaskIndex, sizeof(int)));

    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

	Mandelbrot0<float><<<grid, threads>>>(out, imageW, imageH, crunch, (float)xOff, (float)yOff, (float)scale, colors, frame, animationFrame, pMatrix, pLight);
    CUT_CHECK_ERROR("Mandelbrot kernel execution failed.\n");

    // copy results from device to host
    CUDA_SAFE_CALL(hipMemcpy(out, d_data, mem_size, hipMemcpyDeviceToHost));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_data));
    CUDA_SAFE_CALL(hipFree(pMatrix));
    CUDA_SAFE_CALL(hipFree(pLight));
} // RunMandelbrot0

#endif