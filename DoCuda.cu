#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cutil4win.h"
#include <d3dx9.h>
#include <atltypes.h>

#define TARGET __device__
#include "task.h"
#include "matrix.h"

struct Info
{
	bool			valid;
	D3DMATERIAL9	Material;
	BOOL			isEnter;	// ���荞��
	double			Distance;	// ��_�܂ł̋���
	sp				Cross;		// ��_���W
	sp				Vertical;	// �@���x�N�g��
	double			Refractive;	// ���ܗ�
	NodeInfo		nodeInfo;	//
};

struct Stack
{
	Info*	data[100];
	int		index;
};

#ifndef M_PI
#define M_PI (4.0*atan(1.0))
#endif

__constant__ Task cTask[100];
__constant__ int  cTaskIndex;
static int sTaskIndex = 0;

__device__
void GetInfo_Sphere(const Task& task, const sp& K, const sp& L, Info& info)
{
	double	a = K * K;
	double	b = K * L;
	double	c = L * L - 1.0; 

	double	bb_ac = b*b - a*c;

	if (bb_ac < 0) {
		info.valid = false;
		return;
	}

	double	t;
	double	t1 = (-b + sqrt(bb_ac)) / a;
	double	t2 = (-b - sqrt(bb_ac)) / a;

	info.isEnter = 0;
	if (t1 > 0) {
		if (t2 > 0) {
			t = (t1 < t2) ? t1 : t2;
			info.isEnter = 1;
		} else
			t = t1;
	} else {
		if (t2 > 0)
			t = t2;
		else {
			info.valid = false;
			return;
		}
	}

	info.Cross = info.Vertical = K * t + L;
	info.Distance = t * sqrt(K * K);

	double x,y,z, th, phy;

	x = info.Vertical.x;
	y = info.Vertical.y;
	z = info.Vertical.z;

	th = atan2(y, sqrt(x*x+z*z)) / M_PI + .5;
	phy = atan2(x, -z) / (2 * M_PI) + .5;

	//info.Material = GetPixel(phy, th).getMaterial();
	info.Material = sp(	256 * task.nodeInfo.m_Material.Diffuse.r,
						256 * task.nodeInfo.m_Material.Diffuse.g,
						256 * task.nodeInfo.m_Material.Diffuse.b).getMaterial();
	info.nodeInfo = task.nodeInfo;
	info.valid = true;
}

__device__
bool GetInfo2(const sp& K, const sp& L, Info& info)
{
	Stack	stack;
	stack.index = 0;
	
	// stack.data �̃������m�ۂ����鎖�B

	
	for (int idx = 0; idx < cTaskIndex; idx++) {
		Info	inf;
		
		matrix m;
		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				m.m_data[i][j] = cTask[idx].m[i][j];
			}
		}
		matrix Inv_m = m.Inv();

		sp L2 = Inv_m * L;
		sp K2 = Inv_m * (K + L) - L2;

		switch (cTask[idx].type) {
		case SPHERE:
			GetInfo_Sphere(cTask[idx], K2, L2, inf);

			// inf ���X�^�b�N�ɐςށB
	
			break;
		case PLANE:
			break;
		case PLUS:
//			GetInfo_Plus(cTask[idx], k, l, inf);
			break;
		case MINUS:
			break;
		case MULTIPLE:
			break;
		case CONE:
			break;
		case CYLINDER:
			break;
		case TORUS:
			break;
		case POLYGON:
			break;
		case CUBE:
			break;
		case TEAPOT:
			break;
		}
	}

	bool ans = stack.data[0]->valid;
	
	// stack.data �̃�������������鎖�B
	
	return ans;
}

__device__
sp GetColor(sp K, sp L, const sp& Light)
{
	sp ans(127,127,127);

	enum eMode {
		eNone, eReflect, eThrough
	};
	eMode mode = eNone;

	Info info;
	
	for (int i = 0; i < 10; i++) {
		
		switch (mode) {
		case eNone:
			break;
		case eReflect:
			// ���˗��ŐF��������B
			info.Material = (info.nodeInfo.m_Reflect * ans + (1 - info.nodeInfo.m_Reflect) * sp(info.Material)).getMaterial();
			break;
		case eThrough:
			// ���ߗ��ŐF��������B
			info.Material = (info.nodeInfo.m_Through * ans + (1 - info.nodeInfo.m_Through) * sp(info.Material)).getMaterial();
			break;
		}

		if (!GetInfo2(K, L, info))
			break;
			
		sp k = K.e();
		sp v = info.Vertical.e();

		// ���˗�������ꍇ�A
		if (info.nodeInfo.m_Reflect > 0) {
			sp k2 = k - 2 * (v * k) * v;
			sp l2 = info.Cross + 1E-05 * k2;
			// ���˂��������x�N�g������F���擾�B
			K = k2, L = l2, mode = eReflect;
			continue;
		}

		// ���ߗ�������ꍇ�A
		if (info.nodeInfo.m_Through > 0) {
			double r = info.Refractive;
			double i = k * v;
			sp k2 = r * (k -i * v - sqrt(r * r - 1.0 + i * i) * v);
			sp l2 = info.Cross + 1E-05 * k2;
			// ���܂��������x�N�g������F���擾�B
			K = k2, L = l2, mode = eThrough;
			continue;
		}

		// �������F��␳�B
		double	x = -Light.e() * info.Vertical.e();
		x = (x > 0.0) ? x : 0.0;
		double t = 64 + 191 * sin(M_PI / 2 * x);
		double b = 191 * (1 - cos(M_PI / 2 * x));
		
		ans =  (t - b) * sp(info.Material) / 255 + sp(b,b,b);
	}

	return ans;
}

__global__
void kernel(unsigned long* dst, int imageW, int imageH, matrix* pMatrix, sp* pLight)
{
    const int px = blockDim.x * blockIdx.x + threadIdx.x;
    const int py = blockDim.y * blockIdx.y + threadIdx.y;
   	double rx = 20.0 * px / imageW - 10.0;
	double ry = 20.0 * py / imageH - 10.0;

	sp k(0.01 * rx / 20.0, 0.01 * ry / 20.0, 0.01);
	sp l(rx, ry, -20);
    
	k = *pMatrix * (k + l) - *pMatrix * l;
	l = *pMatrix * l;

	sp c = GetColor(k, l, *pLight);
//	sp c((imageW * cTaskIndex) % 256, (imageH * cTaskIndex) % 256, 0);

	if (px <= imageW && py <= imageH) {
		dst[px + py * imageW] = RGB(c.x, c.y, c.z);
	}
}

void DoCuda(unsigned long* out, const int imageW, const int imageH, const matrix* m, const sp* light)
{
	matrix*		pMatrix;
	sp*			pLight;
	unsigned long* d_data;
    unsigned int mem_size = imageW * imageH * sizeof(unsigned long);
    

    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, mem_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&pMatrix, sizeof(matrix)));
	CUDA_SAFE_CALL(hipMalloc((void**)&pLight, sizeof(sp)));

	CUDA_SAFE_CALL(hipMemcpy(pMatrix, m, sizeof(matrix), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(pLight, light, sizeof(sp), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol<int>(cTaskIndex, &sTaskIndex, sizeof(int)));

    dim3 threads(16,16);
    dim3 grid(16,16);
//    dim3 grid(iDivUp(imageW, 16), iDivUp(imageH, 16));

	// execute the kernel
	kernel<<< grid, threads >>>(d_data, imageW, imageH, pMatrix, pLight);
	CUT_CHECK_ERROR("kernel failed.");

    // copy results from device to host
    CUDA_SAFE_CALL(hipMemcpy(out, d_data, mem_size, hipMemcpyDeviceToHost));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_data));
    CUDA_SAFE_CALL(hipFree(pMatrix));
    CUDA_SAFE_CALL(hipFree(pLight));
}

void ClearTask()
{
	sTaskIndex = 0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol<int>(cTaskIndex, &sTaskIndex, sizeof(int)));
}

void AddTask(const Task& task)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol<Task>(cTask[0], &task, sizeof(Task), sTaskIndex * sizeof(Task)));
	sTaskIndex++;
}
