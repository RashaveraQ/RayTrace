#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cutil4win.h"
#include <d3dx9.h>
#include <atltypes.h>

#define TARGET __device__
#include "task.h"
#include "sp.h"
#include "matrix.h"

#ifndef M_PI
#define M_PI (4.0*atan(1.0))
#endif

__constant__ Task cTask[100];
__constant__ int  cTaskIndex;
static int sTaskIndex = 0;

struct Info4cuda
{
	bool			valid;
	D3DMATERIAL9	Material;
	BOOL			isEnter;	// ���荞��
	double			Distance;	// ��_�܂ł̋���
	double			Cross_x;	// ��_���Wx
	double			Cross_y;	// ��_���Wy
	double			Cross_z;	// ��_���Wz
	double			Vertical_x;	// �@���x�N�g��x
	double			Vertical_y;	// �@���x�N�g��y
	double			Vertical_z;	// �@���x�N�g��z
	double			Refractive;	// ���ܗ�
	NodeInfo		nodeInfo;	//
};

#define STACK_SIZE	20

struct Stack {
	int				Index;
	bool			valid		[STACK_SIZE];
	D3DMATERIAL9	Material	[STACK_SIZE];
	BOOL			isEnter		[STACK_SIZE];	// ���荞��
	double			Distance	[STACK_SIZE];	// ��_�܂ł̋���
	double			Cross_x		[STACK_SIZE];	// ��_���Wx
	double			Cross_y		[STACK_SIZE];	// ��_���Wy
	double			Cross_z		[STACK_SIZE];	// ��_���Wz
	double			Vertical_x	[STACK_SIZE];	// �@���x�N�g��x
	double			Vertical_y	[STACK_SIZE];	// �@���x�N�g��y
	double			Vertical_z	[STACK_SIZE];	// �@���x�N�g��z
	double			Refractive	[STACK_SIZE];	// ���ܗ�
	NodeInfo		nodeInfo	[STACK_SIZE];	//
};

__device__
void GetInfo_Sphere(const Task& task, const sp& K, const sp& L, Info4cuda& info)
{
	double	a = K * K;
	double	b = K * L;
	double	c = L * L - 1.0; 

	double	bb_ac = b*b - a*c;

	if (bb_ac < 0) {
		info.valid = false;
		return;
	}

	double	t;
	double	t1 = (-b + sqrt(bb_ac)) / a;
	double	t2 = (-b - sqrt(bb_ac)) / a;

	info.isEnter = 0;
	if (t1 > 0) {
		if (t2 > 0) {
			t = (t1 < t2) ? t1 : t2;
			info.isEnter = 1;
		} else
			t = t1;
	} else {
		if (t2 > 0)
			t = t2;
		else {
			info.valid = false;
			return;
		}
	}

	sp cross = K * t + L;
	info.Cross_x = info.Vertical_x = cross.x;
	info.Cross_y = info.Vertical_y = cross.y;
	info.Cross_z = info.Vertical_z = cross.z;
	
	info.Distance = t * sqrt(K * K);

	double x,y,z, th, phy;

	x = info.Vertical_x;
	y = info.Vertical_y;
	z = info.Vertical_z;

	th = atan2(y, sqrt(x*x+z*z)) / M_PI + .5;
	phy = atan2(x, -z) / (2 * M_PI) + .5;

	//info.Material = GetPixel(phy, th).getMaterial();
	info.Material = sp(	256 * task.nodeInfo.m_Material.Diffuse.r,
						256 * task.nodeInfo.m_Material.Diffuse.g,
						256 * task.nodeInfo.m_Material.Diffuse.b).getMaterial();
	info.nodeInfo = task.nodeInfo;
	info.valid = true;
}

__device__
void GetInfo_Cone(const Task& task, const sp& K, const sp& L, Info4cuda& info)
{
	if (L.y > 1) {
		if (K.y >= 0) {
			info.valid = false;
			return;
		}
		double t = (1 - L.y) / K.y;
		sp p = K*t + L;
		if (p.x * p.x + p.z * p.z <= 1) {
			info.Cross_x = p.x;
			info.Cross_y = p.y;
			info.Cross_z = p.z;
			info.Vertical_x = 0;
			info.Vertical_y = 1;
			info.Vertical_z = 0;
			info.Distance = t * sqrt(K*K);
			info.isEnter = 1;
//			info.Material = GetPixel(.5*(p.x+1),.5*(p.z+1)).getMaterial();
			info.Material = sp(256 * task.nodeInfo.m_Material.Diffuse.r,
							   256 * task.nodeInfo.m_Material.Diffuse.g,
							   256 * task.nodeInfo.m_Material.Diffuse.b).getMaterial();
			info.nodeInfo = task.nodeInfo;
			info.Refractive = task.nodeInfo.m_Refractive;
			info.valid = true;
			return;
		}
	}

	double	a, b, c, d, t, t1, t2;
	c = K.x * L.y - K.y * L.x, c *= c, d = c;
	c = K.z * L.y - K.y * L.z, c *= c, d += c;
	c = K.x * L.z - K.z * L.x, c *= c, d -= c;
	if ( d < 0 ) {
		info.valid = false;
		return;
	}
	d = sqrt( d );
	a = -( K.x * L.x + K.z * L.z - K.y * L.y );
	b = K.x * K.x + K.z * K.z - K.y * K.y;
	t1 = ( a + d ) / b;
	t2 = ( a - d ) / b;
	if (fabs(t1) < 1E-10 || fabs(t2) < 1E-10) {
		info.valid = false;
		return;
	}
	if (t1 > 0) {
		if (t2 > 0) {
			if (t1 < t2)
				t = (K.y * t1 + L.y > 0) ? t1 : t2;
			else
				t = (K.y * t2 + L.y > 0) ? t2 : t1;
		} else
			t = t1;
	} else {
		if (t2 > 0)
			t = t2;
		else {
			info.valid = false;
			return;
		}
	}

	info.isEnter = (0 > L.y || L.y > 1 || sqrt(L.x * L.x + L.z * L.z) > L.y);
	sp p = K * t + L;
	info.Cross_x = info.Vertical_x = p.x;
	info.Cross_y = info.Vertical_y = p.y;
	info.Cross_z = info.Vertical_z = p.z;

	if (p.y < 0 || p.x * p.x + p.z * p.z > 1) {
		info.valid = false;
		return;
	}

	info.Vertical_y *= -1;

	info.Distance = t * sqrt(K * K);
	info.Material = task.nodeInfo.m_Material;
	info.nodeInfo = task.nodeInfo;
	info.valid = true;
}

__device__
void GetInfo_Cylinder(const Task& task, const sp& K, const sp& L, Info4cuda& info)
{
	if (L.y < -1) {
		if (K.y <= 0) {
			info.valid = false;
			return;
		}

		double t = -(1 + L.y) / K.y;
		sp	p = K * t + L;

		if (p.x * p.x + p.z * p.z <= 1) {
			info.Cross_x = p.x;
			info.Cross_y = p.y;
			info.Cross_z = p.z;
			info.Vertical_x = 0;
			info.Vertical_y = -1;
			info.Vertical_z = 0;
			info.Distance = t * sqrt(K * K);
			info.isEnter = 1;
//			info.Material = GetPixel(.5*(p.x+1),.5*(p.z+1)).getMaterial();
			info.Material = sp(256 * task.nodeInfo.m_Material.Diffuse.r,
							   256 * task.nodeInfo.m_Material.Diffuse.g,
							   256 * task.nodeInfo.m_Material.Diffuse.b).getMaterial();
			info.nodeInfo = task.nodeInfo;
			info.valid = true;
			return;
		}
	}

	if (L.y > 1) {
		if (K.y >= 0) {
			info.valid = false;
			return;
		}

		double t = (1 - L.y) / K.y;

		sp	p = K*t+L;

		if (p.x * p.x + p.z * p.z <= 1) {
			info.Cross_x = p.x;
			info.Cross_y = p.y;
			info.Cross_z = p.z;
			info.Vertical_x = 0;
			info.Vertical_y = 1;
			info.Vertical_z = 0;
			info.Distance = t * sqrt(K * K);
			info.isEnter = 1;
//			info.Material = GetPixel(.5*(p.x+1),.5*(p.z+1)).getMaterial();
			info.Material = sp(256 * task.nodeInfo.m_Material.Diffuse.r,
							   256 * task.nodeInfo.m_Material.Diffuse.g,
							   256 * task.nodeInfo.m_Material.Diffuse.b).getMaterial();
			info.nodeInfo = task.nodeInfo;
			info.valid = true;
			return;
		}
	}

	double	a, b, c, d, t, t1, t2;

	c = K.x * L.z - K.z * L.x;
	c *= c;
	a = K.x * K.x + K.z * K.z;
	d = a - c;

	if (d < 0) {
		info.valid = false;
		return;
	}

	d = sqrt(d);
	b = -(K.x * L.x + K.z * L.z);

	t1 = (b + d) / a;
	t2 = (b - d) / a;

	info.isEnter = 0;
	if (t1 > 0) {
		if (t2 > 0) {
			t = (t1 < t2) ? t1 : t2;
			info.isEnter = 1;
		} else
			t = t1;
	} else {
		if (t2 > 0)
			t = t2;
		else {
			info.valid = false;
			return;
		}
	}

	sp p = K * t + L;

	if (p.y < -1 || 1 < p.y) {
		info.valid = false;
		return;
	}

	info.isEnter = (-1 > L.y || L.y > 1 || sqrt(L.x*L.x+L.z*L.z) > 1.0);

	info.Cross_x = info.Vertical_x = p.x;
	info.Cross_y = info.Vertical_y = p.y;
	info.Cross_z = info.Vertical_z = p.z;
	info.Vertical_y = 0;
	info.Distance = t * sqrt(K * K);
	info.Material = task.nodeInfo.m_Material;
	info.nodeInfo = task.nodeInfo;
	info.valid = true;
}

#include "Torus4cuda.cu"

__device__
void GetInfo_Cube(const Task& task, const sp& K, const sp& L, Info4cuda& info)
{
	info.isEnter = (-1.0 > L.x || L.x > 1.0 || -1.0 > L.y || L.y > 1.0 || -1.0 > L.z || L.z > 1.0);

	if (info.isEnter) {
		//�O������O�Ɍ������ꍇ�A
		if ((L.x < -1.0 && K.x <= 0.0) || (1.0 < L.x && 0.0 <= K.x) ||
			(L.y < -1.0 && K.y <= 0.0) || (1.0 < L.y && 0.0 <= K.y) ||
			(L.z < -1.0 && K.z <= 0.0) || (1.0 < L.z && 0.0 <= K.z)) {
			info.valid = false;
			return;
		}
	}

	int		i, j, k;
	double	T[6], t1, t;
	sp		c;

	// �S�Ă̖ʂ܂ł̋��������߂�B
	T[0] = (1.0 - L.x) / K.x; T[1] = - (1.0 + L.x) / K.x;
	T[2] = (1.0 - L.y) / K.y; T[3] = - (1.0 + L.y) / K.y;
	T[4] = (1.0 - L.z) / K.z; T[5] = - (1.0 + L.z) / K.z;

	// t1 �ɍő�l��������B
	for (i = 1, t1 = T[0]; i < 6; i++)
		if (t1 < T[i])
			t1 = T[i];

	// ���̏ꍇ�At1 ��������B
	for (i = 0; i < 6; i++)
		if (T[i] < 0.0)
			T[i] = t1;

	for (k = 0; k < 3; k++)	{
		for (i = 1, j = 0, t= T[0]; i < 6; i++)
			if (t > T[i])
				t = T[i], j = i;

		if (!info.isEnter)
			break;

		c = K * t + L;

		if (-1.0 <= c.x && c.x <= 1.0 && -1.0 <= c.y && c.y <= 1.0 && -1.0 <= c.z && c.z <= 1.0)
			break;

		T[j] = t1;
	}

	if (k == 3) {
		info.valid = false;
		return;
	}

	sp cross = K * t + L;
	info.Cross_x = cross.x;
	info.Cross_y = cross.y;
	info.Cross_z = cross.z;
	info.Distance = t * sqrt(K * K);
	info.Material = task.nodeInfo.m_Material;

	switch (j) {
	case 0:
		info.Vertical_x = 1;
		info.Vertical_y = 0;
		info.Vertical_z = 0;
		break;
	case 1:
		info.Vertical_x = -1;
		info.Vertical_y = 0;
		info.Vertical_z = 0;
		break;
	case 2:
		info.Vertical_x = 0;
		info.Vertical_y = 1;
		info.Vertical_z = 0;
		break;
	case 3:
		info.Vertical_x = 0;
		info.Vertical_y = -1;
		info.Vertical_z = 0;
		break;
	case 4:
		info.Vertical_x = 0;
		info.Vertical_y = 0;
		info.Vertical_z = 1;
		break;
	case 5:
		info.Vertical_x = 0;
		info.Vertical_y = 0;
		info.Vertical_z = -1;
		break;
	}
	info.nodeInfo = task.nodeInfo;
	info.valid = true;
}

__device__
void GetInfo_Plus(const Task& task, const sp& K, const sp& L, Info4cuda& info, Stack& stack)
{
	Info4cuda	tmp;
	double	l = -1;

	for (int i = 0; i < task.data.gathering.member; i++) {
		int j = --stack.Index;
		if (stack.valid[j]) {
			if (l == -1 || stack.Distance[j] < l) {
				l = stack.Distance[j];
				info.valid		= stack.valid	[j];
				info.Material	= stack.Material[j];
				info.isEnter	= stack.isEnter	[j];	// ���荞��
				info.Distance	= stack.Distance[j];	// ��_�܂ł̋���
				info.Cross_x	= stack.Cross_x[j];		// ��_���Wx
				info.Cross_y	= stack.Cross_y[j];		// ��_���Wy
				info.Cross_z	= stack.Cross_z[j];		// ��_���Wz
				info.Vertical_x	= stack.Vertical_x[j];	// �@���x�N�g��x
				info.Vertical_y	= stack.Vertical_y[j];	// �@���x�N�g��y
				info.Vertical_z	= stack.Vertical_z[j];	// �@���x�N�g��z
				info.Refractive	= stack.Refractive[j];	// ���ܗ�
				info.nodeInfo	= stack.nodeInfo[j];	//
			}
		}
	}

	if (l < 0) {
		info.valid = false;
		return;
	}

	if (info.Material.Diffuse.r < 0)
		info.Material = task.nodeInfo.m_Material;
	info.valid = true;
}

__device__
bool GetInfo2(const sp& K, const sp& L, Info4cuda& info)
{
	Stack stack;
	stack.Index = 0;
	
	for (int idx = 0; idx < cTaskIndex; idx++) {
		Info4cuda	inf;
		
		matrix m;
		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				m.m_data[i][j] = cTask[idx].m[i][j];
			}
		}

		sp L2 = m * L;
		sp K2 = m * (K + L) - L2;

		switch (cTask[idx].type) {
		case SPHERE:
			GetInfo_Sphere(cTask[idx], K2, L2, inf);
			break;
		case PLANE:
			break;
		case PLUS:
			GetInfo_Plus(cTask[idx], K2, L2, inf, stack);
			break;
		case MINUS:
			break;
		case MULTIPLE:
			break;
		case CONE:
			GetInfo_Cone(cTask[idx], K2, L2, inf);
			break;
		case CYLINDER:
			GetInfo_Cylinder(cTask[idx], K2, L2, inf);
			break;
		case TORUS:
			GetInfo_Torus(cTask[idx], K2, L2, inf);
			break;
		case POLYGON:
			break;
		case CUBE:
			GetInfo_Cube(cTask[idx], K2, L2, inf);
			break;
		case TEAPOT:
			break;
		default:
			break;
		}
		
		sp vertical(inf.Vertical_x, inf.Vertical_y, inf.Vertical_z);
		sp cross(inf.Cross_x, inf.Cross_y, inf.Cross_z);
		
		matrix Inv_m = m.Inv();
		vertical = Inv_m * (vertical + cross) - Inv_m * cross;
		cross = Inv_m * cross;
		inf.Distance = (cross - L).abs();
		inf.Refractive = inf.nodeInfo.m_Refractive;
		if (inf.isEnter)
			inf.Refractive = 1 / inf.Refractive;

		inf.Vertical_x = vertical.x;
		inf.Vertical_y = vertical.y;
		inf.Vertical_z = vertical.z;
		inf.Cross_x = cross.x;
		inf.Cross_y = cross.y;
		inf.Cross_z = cross.z;
		
		// inf ���X�^�b�N�ɐςށB
		stack.valid			[stack.Index] = inf.valid;
		stack.Material		[stack.Index] = inf.Material;
		stack.isEnter		[stack.Index] = inf.isEnter;	// ���荞��
		stack.Distance		[stack.Index] = inf.Distance;	// ��_�܂ł̋���
		stack.Cross_x		[stack.Index] = inf.Cross_x;	// ��_���Wx
		stack.Cross_y		[stack.Index] = inf.Cross_y;	// ��_���Wy
		stack.Cross_z		[stack.Index] = inf.Cross_z;	// ��_���Wz
		stack.Vertical_x	[stack.Index] = inf.Vertical_x;	// �@���x�N�g��x
		stack.Vertical_y	[stack.Index] = inf.Vertical_y;	// �@���x�N�g��y
		stack.Vertical_z	[stack.Index] = inf.Vertical_z;	// �@���x�N�g��z
		stack.Refractive	[stack.Index] = inf.Refractive;	// ���ܗ�
		stack.nodeInfo		[stack.Index] = inf.nodeInfo;	//
		stack.Index++;
		info = inf;
	}

	return info.valid;
}

__device__
sp GetColor(sp K, sp L, const sp& Light)
{
	sp ans(127,127,127);

	enum eMode {
		eNone, eReflect, eThrough
	};
	eMode mode = eNone;

	Info4cuda info;
	
	for (int i = 0; i < 10; i++) {
		
		switch (mode) {
		case eNone:
			break;
		case eReflect:
			// ���˗��ŐF��������B
			info.Material = (info.nodeInfo.m_Reflect * ans + (1 - info.nodeInfo.m_Reflect) * sp(info.Material)).getMaterial();
			break;
		case eThrough:
			// ���ߗ��ŐF��������B
			info.Material = (info.nodeInfo.m_Through * ans + (1 - info.nodeInfo.m_Through) * sp(info.Material)).getMaterial();
			break;
		}

		if (!GetInfo2(K, L, info))
			break;
			
		sp k = K.e();
		sp v(info.Vertical_x, info.Vertical_y, info.Vertical_y);

		v = v.e();
/*
		// ���˗�������ꍇ�A
		if (info.nodeInfo.m_Reflect > 0) {
			sp k2 = k - 2 * (v * k) * v;
			sp cross(info.Cross_x, info.Cross_y, info.Cross_z);
			sp l2 = cross + 1E-05 * k2;
			// ���˂��������x�N�g������F���擾�B
			K = k2, L = l2, mode = eReflect;
			continue;
		}

		// ���ߗ�������ꍇ�A
		if (info.nodeInfo.m_Through > 0) {
			double r = info.Refractive;
			double i = k * v;
			sp k2 = r * (k -i * v - sqrt(r * r - 1.0 + i * i) * v);
			sp cross(info.Cross_x, info.Cross_y, info.Cross_z);
			sp l2 = cross + 1E-05 * k2;
			// ���܂��������x�N�g������F���擾�B
			K = k2, L = l2, mode = eThrough;
			continue;
		}
*/
		// �������F��␳�B
		sp vertical(info.Vertical_x, info.Vertical_y, info.Vertical_z);
		
		double	x = -Light.e() * vertical.e();
		x = (x > 0.0) ? x : 0.0;
		double t = 64 + 191 * sin(M_PI / 2 * x);
		double b = 191 * (1 - cos(M_PI / 2 * x));
		
		ans =  (t - b) * sp(info.Material) / 255 + sp(b,b,b);
	}

	return ans;
}

__global__
void kernel(unsigned long* dst, int imageW, int imageH, matrix* pMatrix, sp* pLight)
{
    const int px = blockDim.x * blockIdx.x + threadIdx.x;
    const int py = blockDim.y * blockIdx.y + threadIdx.y;
   	double rx = 20.0 * px / imageW - 10.0;
	double ry = 20.0 * py / imageH - 10.0;

	sp k(0.01 * rx / 20.0, 0.01 * ry / 20.0, 0.01);
	sp l(rx, ry, -20);
    
	k = *pMatrix * (k + l) - *pMatrix * l;
	l = *pMatrix * l;

	sp c = GetColor(k, l, *pLight);

	if (px <= imageW && py <= imageH) {
		dst[px + py * imageW] = RGB(c.x, c.y, c.z);
	}
}

void DoCuda(unsigned long* out, const int imageW, const int imageH, const matrix* m, const sp* light)
{
	matrix*		pMatrix;
	sp*			pLight;
	unsigned long* d_data;
    unsigned int mem_size = imageW * imageH * sizeof(unsigned long);
    

    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, mem_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&pMatrix, sizeof(matrix)));
	CUDA_SAFE_CALL(hipMalloc((void**)&pLight, sizeof(sp)));

	CUDA_SAFE_CALL(hipMemcpy(pMatrix, m, sizeof(matrix), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(pLight, light, sizeof(sp), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol<int>(cTaskIndex, &sTaskIndex, sizeof(int)));

    dim3 threads(1,1);
    dim3 grid(488,488);
//    dim3 grid(iDivUp(imageW, 16), iDivUp(imageH, 16));

	// execute the kernel
	kernel<<< grid, threads >>>(d_data, imageW, imageH, pMatrix, pLight);
	CUT_CHECK_ERROR("kernel failed.");

    // copy results from device to host
    CUDA_SAFE_CALL(hipMemcpy(out, d_data, mem_size, hipMemcpyDeviceToHost));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_data));
    CUDA_SAFE_CALL(hipFree(pMatrix));
    CUDA_SAFE_CALL(hipFree(pLight));
}

void ClearTask()
{
	sTaskIndex = 0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol<int>(cTaskIndex, &sTaskIndex, sizeof(int)));
}

void AddTask(const Task& task)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol<Task>(cTask[0], &task, sizeof(Task), sTaskIndex * sizeof(Task)));
	sTaskIndex++;
}
