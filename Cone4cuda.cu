#include "hip/hip_runtime.h"
__device__
void GetInfo_Cone(const Task& task, const sp& K, const sp& L, Info4cuda& info)
{
	if (L.y > 1) {
		if (K.y >= 0) {
			info.valid = false;
			return;
		}
		float t = (1 - L.y) / K.y;
		sp p = K*t + L;
		if (p.x * p.x + p.z * p.z <= 1) {
			info.Cross_x = p.x;
			info.Cross_y = p.y;
			info.Cross_z = p.z;
			info.Vertical_x = 0;
			info.Vertical_y = 1;
			info.Vertical_z = 0;
			info.Distance = t * sqrt(K*K);
			info.isEnter = 1;
//			info.Material = GetPixel(.5*(p.x+1),.5*(p.z+1)).getMaterial();
			info.Material = sp(256 * task.nodeInfo.m_Material.Diffuse.r,
							   256 * task.nodeInfo.m_Material.Diffuse.g,
							   256 * task.nodeInfo.m_Material.Diffuse.b).getMaterial();
			info.nodeInfo = task.nodeInfo;
			info.Refractive = task.nodeInfo.m_Refractive;
			info.valid = true;
			return;
		}
	}

	float	a, b, c, d, t, t1, t2;
	c = K.x * L.y - K.y * L.x, c *= c, d = c;
	c = K.z * L.y - K.y * L.z, c *= c, d += c;
	c = K.x * L.z - K.z * L.x, c *= c, d -= c;
	if ( d < 0 ) {
		info.valid = false;
		return;
	}
	d = sqrt( d );
	a = -( K.x * L.x + K.z * L.z - K.y * L.y );
	b = K.x * K.x + K.z * K.z - K.y * K.y;
	t1 = ( a + d ) / b;
	t2 = ( a - d ) / b;
	if (fabs(t1) < 1E-10 || fabs(t2) < 1E-10) {
		info.valid = false;
		return;
	}
	if (t1 > 0) {
		if (t2 > 0) {
			if (t1 < t2)
				t = (K.y * t1 + L.y > 0) ? t1 : t2;
			else
				t = (K.y * t2 + L.y > 0) ? t2 : t1;
		} else
			t = t1;
	} else {
		if (t2 > 0)
			t = t2;
		else {
			info.valid = false;
			return;
		}
	}

	info.isEnter = (0 > L.y || L.y > 1 || sqrt(L.x * L.x + L.z * L.z) > L.y);
	sp p = K * t + L;
	info.Cross_x = info.Vertical_x = p.x;
	info.Cross_y = info.Vertical_y = p.y;
	info.Cross_z = info.Vertical_z = p.z;

	if (p.y < 0 || p.x * p.x + p.z * p.z > 1) {
		info.valid = false;
		return;
	}

	info.Vertical_y *= -1;

	info.Distance = t * sqrt(K * K);
	info.Material = task.nodeInfo.m_Material;
	info.nodeInfo = task.nodeInfo;
	info.valid = true;
}
